#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>

#include <cmath>
#include <cstdio>  // printf
#include <iostream>
#include <memory>
#include <mutex>
#include <stdexcept>
#include <vector>

#include "cudaOP.cuh"
#include "cutlass/cutlass.h"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"

#define WARP_SIZE 32

inline void checkCublasStatus(hipblasStatus_t status, const char *file, int line) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        char errorMsg[256];
        // Note: cublasGetErrorString is not a standard function.
        // Provide a basic message.
        snprintf(errorMsg, sizeof(errorMsg), "cuBLAS error %d at %s:%d", static_cast<int>(status), file, line);
        fprintf(stderr, "%s\n", errorMsg);
        throw std::runtime_error(errorMsg);
    }
}
#define CHECK_CUBLAS(call) checkCublasStatus(call, __FILE__, __LINE__)

namespace cuda_OP {
// === Warp Reduce Sum 模板函数 ===
template <typename T, const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ T warp_reduce_sum(T val) {
#pragma unroll
    for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
        val += __shfl_down_sync(0xffffffff, val, mask);
    }
    return val;
}

// === 高效的 GEMV kernel (M=1, 支持bias) ===
// 计算 C = A * B^T + bias，其中 A: [1, K], B: [N, K], C: [1, N]
template <typename T>
__global__ void gemv_with_bias_kernel(const T *A, const T *B, const T *bias, T *C, int M, int K, int N) {
    // A: [1, K] (行主序), B: [N, K] (列主序存储！), C: [1, N]
    // 计算: C[n] = sum(A[k] * B[n, k]) + bias[n] for k in [0, K)
    // 注意：B是列主序存储，所以 B[n, k] = B[n * K + k]
    // 每个warp负责一个输出元素
    int tx = threadIdx.x;          // 0~31
    int ty = threadIdx.y;          // 0~blockDim.y
    int bx = blockIdx.x;           // 0~(N-1)/blockDim.y
    int lane = tx % WARP_SIZE;     // 0~31
    int n = bx * blockDim.y + ty;  // 输出元素索引

    if (n < N) {
        T sum = T(0);

        // 计算需要的warp数量来覆盖K维度
        int NUM_WARPS = (K + WARP_SIZE - 1) / WARP_SIZE;

#pragma unroll
        for (int w = 0; w < NUM_WARPS; ++w) {
            int k = w * WARP_SIZE + lane;
            if (k < K) {
                // A[k] * B[n, k]，B是列主序存储，所以 B[n, k] = B[n * K + k]
                sum += A[k] * B[n * K + k];
            }
        }

        // warp内规约
        sum = warp_reduce_sum<T, WARP_SIZE>(sum);

        // 只有第一个线程写结果，并加上bias
        if (lane == 0) {
            C[n] = sum + bias[n];
        }
    }
}

// === 高效的 GEMV kernel (M=1, 无bias) ===
// 计算 C = A * B^T，其中 A: [1, K], B: [N, K], C: [1, N]
template <typename T>
__global__ void gemv_kernel(const T *A, const T *B, T *C, int M, int K, int N) {
    // A: [1, K] (行主序), B: [N, K] (列主序存储！), C: [1, N]
    // 计算: C[n] = sum(A[k] * B[n, k]) for k in [0, K)
    // 注意：B是列主序存储，所以 B[n, k] = B[n * K + k]
    // 每个warp负责一个输出元素
    int tx = threadIdx.x;          // 0~31
    int ty = threadIdx.y;          // 0~blockDim.y
    int bx = blockIdx.x;           // 0~(N-1)/blockDim.y
    int lane = tx % WARP_SIZE;     // 0~31
    int n = bx * blockDim.y + ty;  // 输出元素索引

    if (n < N) {
        T sum = T(0);

        // 计算需要的warp数量来覆盖K维度
        int NUM_WARPS = (K + WARP_SIZE - 1) / WARP_SIZE;

#pragma unroll
        for (int w = 0; w < NUM_WARPS; ++w) {
            int k = w * WARP_SIZE + lane;
            if (k < K) {
                // A[k] * B[n, k]，B是列主序存储，所以 B[n, k] = B[n * K + k]
                sum += A[k] * B[n * K + k];
            }
        }

        // warp内规约
        sum = warp_reduce_sum<T, WARP_SIZE>(sum);

        // 只有第一个线程写结果
        if (lane == 0) {
            C[n] = sum;
        }
    }
}

// === 向量化版本的 GEMV kernel (适用于 K 是 4 的倍数) ===
// 计算 C = A * B^T + bias，其中 A: [1, K], B: [N, K], C: [1, N]
template <typename T>
__global__ void gemv_with_bias_vectorized_kernel(const T *A, const T *B, const T *bias, T *C, int M, int K, int N) {
    // 每个线程处理4个K元素来提高内存带宽利用率
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int lane = tx % WARP_SIZE;
    int n = bx * blockDim.y + ty;

    if (n < N) {
        T sum = T(0);

        // 每个warp处理 4*WARP_SIZE 个K元素
        int NUM_WARPS = (((K + WARP_SIZE - 1) / WARP_SIZE) + 4 - 1) / 4;

#pragma unroll
        for (int w = 0; w < NUM_WARPS; ++w) {
            int k_base = (w * WARP_SIZE + lane) * 4;
            // 实际就是4
            constexpr int VEC_UNIT = sizeof(float2) / sizeof(T);
            Vec_2<T, VEC_UNIT> va, vb;

            va.f2 = *reinterpret_cast<const float2 *>(&A[k_base]);
            vb.f2 = *reinterpret_cast<const float2 *>(&B[n * K + k_base]);
            // 向量化加载和计算
            for (int i = 0; i < VEC_UNIT; ++i) {
                sum += static_cast<float>(va.t[i]) * static_cast<float>(vb.t[i]);
            }
        }

        sum = warp_reduce_sum<float, WARP_SIZE>(sum);

        if (lane == 0) {
            C[n] = static_cast<T>(sum) + bias[n];
        }
    }
}

// === 向量化版本的 GEMV kernel (无bias) ===
// 计算 C = A * B^T，其中 A: [1, K], B: [N, K], C: [1, N]
template <typename T>
__global__ void gemv_vectorized_kernel(const T *A, const T *B, T *C, int M, int K, int N) {
    // 每个线程处理4个K元素来提高内存带宽利用率
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int lane = tx % WARP_SIZE;
    int n = bx * blockDim.y + ty;

    if (n < N) {
        float sum = 0.f;

        // 每个warp处理 4*WARP_SIZE 个K元素
        int NUM_WARPS = (((K + WARP_SIZE - 1) / WARP_SIZE) + 4 - 1) / 4;

#pragma unroll
        for (int w = 0; w < NUM_WARPS; ++w) {
            int k_base = (w * WARP_SIZE + lane) * 4;
            // 实际就是4
            constexpr int VEC_UNIT = sizeof(float2) / sizeof(T);
            Vec_2<T, VEC_UNIT> va, vb;

            va.f2 = *reinterpret_cast<const float2 *>(&A[k_base]);
            vb.f2 = *reinterpret_cast<const float2 *>(&B[n * K + k_base]);
            // 向量化加载和计算
            for (int i = 0; i < VEC_UNIT; ++i) {
                sum += static_cast<float>(va.t[i]) * static_cast<float>(vb.t[i]);
            }
        }

        sum = warp_reduce_sum<float, WARP_SIZE>(sum);

        if (lane == 0) {
            C[n] = static_cast<T>(sum);
        }
    }
}

// === 类型映射特化: 将 __hip_bfloat16 转为 cutlass::bfloat16_t ===
// 定义类型转换 traits（可扩展支持更多类型）
template <typename T>
struct to_cutlass_type {
    using type = T;
};
template <>
struct to_cutlass_type<__hip_bfloat16> {
    using type = cutlass::bfloat16_t;  // 专门处理 bfloat16 类型
};

// === 通用 CUTLASS GEMM 调用模板 ===
template <typename ElementA, typename ElementB, typename ElementOutput, typename LayoutA, typename LayoutB,
          typename LayoutOutput, typename ElementAccumulator = float,
          typename ElementComputeEpilogue = ElementAccumulator, typename MMAOp = cutlass::arch::OpClassTensorOp,
          typename SmArch = cutlass::arch::Sm80, typename ShapeMMAThreadBlock = cutlass::gemm::GemmShape<128, 128, 32>,
          typename ShapeMMAWarp = cutlass::gemm::GemmShape<64, 64, 32>,
          typename ShapeMMAOp = cutlass::gemm::GemmShape<16, 8, 8>,
          typename SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, int NumStages = 2>
cutlass::Status run_cutlass_gemm_raw_templated(int m, int n, int k, ElementA const *d_a, ElementB const *d_b,
                                               ElementOutput const *d_bias, ElementOutput *d_d, hipStream_t stream = 0,
                                               ElementComputeEpilogue alpha = ElementComputeEpilogue(1),
                                               int split_k_slices = 1) {
    // 1. 类型转换: 使用 to_cutlass_type 将用户类型映射为 Cutlass 支持类型
    using ElementA_t = typename to_cutlass_type<ElementA>::type;
    using ElementB_t = typename to_cutlass_type<ElementB>::type;
    using ElementOutput_t = typename to_cutlass_type<ElementOutput>::type;

    // 2. 定义 Epilogue 操作: alpha * (A*B) + bias, 不启用 Beta 缩放
    using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
        ElementOutput_t, 128 / cutlass::sizeof_bits<ElementOutput_t>::value, ElementAccumulator, ElementComputeEpilogue,
        cutlass::epilogue::thread::ScaleType::NoBetaScaling>;

    // 3. 定义 GEMM 类型: 指定所有核心模板参数
    using Gemm =
        cutlass::gemm::device::Gemm<ElementA_t, LayoutA, ElementB_t, LayoutB, ElementOutput_t, LayoutOutput,
                                    ElementAccumulator, MMAOp, SmArch, ShapeMMAThreadBlock, ShapeMMAWarp, ShapeMMAOp,
                                    EpilogueOp, SwizzleThreadBlock, NumStages, 8, 8  // 可选的线程副本分区参数
                                    >;

    // 4. 构造问题规模
    cutlass::gemm::GemmCoord problem_size(m, n, k);

    // 5. 构造 TensorRef: 将原始指针和布局转换为 Cutlass 张量引用
    cutlass::TensorRef<ElementA_t, LayoutA> ref_A(const_cast<ElementA_t *>(reinterpret_cast<const ElementA_t *>(d_a)),
                                                  LayoutA(k)  // leading dimension = k
    );
    cutlass::TensorRef<ElementB_t, LayoutB> ref_B(const_cast<ElementB_t *>(reinterpret_cast<const ElementB_t *>(d_b)),
                                                  LayoutB(n));
    cutlass::TensorRef<ElementOutput_t, LayoutOutput> ref_D(reinterpret_cast<ElementOutput_t *>(d_d), LayoutOutput(n));

    // 6. 构造参数对象: 包含输入、输出、bias、alpha、split-K 切片等
    typename Gemm::Arguments arguments{
        problem_size,  ref_A,   ref_B, {reinterpret_cast<const ElementOutput_t *>(d_bias), 0},  // bias ptr + stride
        ref_D,         {alpha},                                                                 // epilogue 参数
        split_k_slices                                                                          // split-K 切片数
    };

    // 7. 分配内部 workspace
    size_t workspace_size = Gemm::get_workspace_size(arguments);
    cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

    // 8. 实例化运算对象，并检查是否可实现
    Gemm gemm_op;
    cutlass::Status status = gemm_op.can_implement(arguments);
    CUTLASS_CHECK(status);

    // 9. 初始化并执行
    status = gemm_op.initialize(arguments, workspace.get());
    CUTLASS_CHECK(status);
    status = gemm_op(stream);  // 在指定的 CUDA 流上执行
    CUTLASS_CHECK(status);

    return status;
}

template <typename T>
__global__ void matmul_kernel(const T *A, const T *B, T *C, int M, int K, int N) {
    __shared__ T As[16][16];
    __shared__ T Bs[16][16];
    int row = blockIdx.y * 16 + threadIdx.y;
    int col = blockIdx.x * 16 + threadIdx.x;
    T sum = T(0);
    // 计算需要的 tile 数量
    int numTiles = (K + 16 - 1) / 16;
    for (int t = 0; t < numTiles; ++t) {
        int A_col = t * 16 + threadIdx.x;
        if (row < M && A_col < K) {
            As[threadIdx.y][threadIdx.x] = A[row * K + A_col];
        } else {
            As[threadIdx.y][threadIdx.x] = T(0);
        }
        int B_row = t * 16 + threadIdx.y;
        if (col < N && B_row < K) {
            Bs[threadIdx.y][threadIdx.x] = B[col * K + B_row];
        } else {
            Bs[threadIdx.y][threadIdx.x] = T(0);
        }
        __syncthreads();
        for (int k = 0; k < 16; ++k) {
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }
        __syncthreads();
        // Use code with caution.
    }
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}
template <typename T>
__global__ void add_bias_kernel(T *C, const T *bias, int M, int N, int ldc) {
    // 计算当前线程负责处理的全局行索引 (row) 和列索引 (col)
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // 边界检查，确保线程在矩阵的有效范围内
    if (row < M && col < N) {
        // 计算 C 矩阵中元素的线性索引 (行主序)
        int c_index = row * ldc + col;

        // 执行加法: C[row][col] = C[row][col] + bias[col]
        C[c_index] = C[c_index] + bias[col];
    }
}
// --------------------------------------------------
// 带偏置的矩阵乘法kernel
// --------------------------------------------------
template <typename T>
__global__ void matmul_with_bias_kernel(const T *A, const T *B, const T *bias, T *C, int M, int K, int N) {
    __shared__ T As[16][16];
    __shared__ T Bs[16][16];
    int row = blockIdx.y * 16 + threadIdx.y;
    int col = blockIdx.x * 16 + threadIdx.x;
    T sum = T(0);
    // 计算需要的 tile 数量
    int numTiles = (K + 16 - 1) / 16;
    for (int t = 0; t < numTiles; ++t) {
        int A_col = t * 16 + threadIdx.x;
        if (row < M && A_col < K) {
            As[threadIdx.y][threadIdx.x] = A[row * K + A_col];
        } else {
            As[threadIdx.y][threadIdx.x] = T(0);
        }
        int B_row = t * 16 + threadIdx.y;
        if (col < N && B_row < K) {
            Bs[threadIdx.y][threadIdx.x] = B[col * K + B_row];
        } else {
            Bs[threadIdx.y][threadIdx.x] = T(0);
        }
        __syncthreads();
        for (int k = 0; k < 16; ++k) {
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }
        __syncthreads();
        // Use code with caution.
    }
    if (row < M && col < N) {
        // 在结果中加上偏置
        C[row * N + col] = sum + bias[col];
    }
}

/**
 * @brief 基于模板的 cuBLAS GEMM 包装函数，输入和输出类型相同。
 *
 * 执行操作：C = alpha * op(A) * op(B) + beta * C
 * 输入矩阵 A、B 以及输出矩阵 C 的类型都由模板参数 InputType 决定
 * (必须是 float 或 hip_bfloat16)。
 * 计算通常在内部使用 FP32 精度执行（由 compute_type
 * 控制），以获得更好的精度和性能。 结果直接写入 d_C 指向的设备内存中。
 *
 * @tparam InputType   输入矩阵 A, B 和输出矩阵 C 的数据类型 (float 或
 * hip_bfloat16)。
 * @param handle        cuBLAS 库句柄。
 * @param transa        指定操作 op(A)：HIPBLAS_OP_N 或 HIPBLAS_OP_T。
 * @param transb        指定操作 op(B)：HIPBLAS_OP_N 或 HIPBLAS_OP_T。
 * @param m             矩阵 op(A) 和矩阵 C 的行数。
 * @param n             矩阵 op(B) 和矩阵 C 的列数。
 * @param k             矩阵 op(A) 的列数和矩阵 op(B) 的行数。
 * @param alpha         用于 op(A) * op(B) 的标量乘子 (指向 const float
 * 的主机指针)。
 * @param d_A           指向设备内存中矩阵 A 的指针 (const InputType*)。
 * @param lda           矩阵 A 的主维度。
 * @param d_B           指向设备内存中矩阵 B 的指针 (const InputType*)。
 * @param ldb           矩阵 B 的主维度。
 * @param beta          用于矩阵 C 的标量乘子 (指向 const float 的主机指针)。
 * @param d_C           指向设备内存中矩阵 C 的指针
 * (InputType*)。结果将写入此处，类型与输入匹配。
 * @param ldc           矩阵 C 的主维度。
 */
template <typename InputType>
void cublas_matmul_wrapper(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n,
                           int k,
                           const float *alpha,  // 改回 const float*
                           const InputType *d_A, int lda, const InputType *d_B, int ldb,
                           const float *beta,  // 改回 const float*
                           InputType *d_C,     // *** 输出类型为 InputType* ***
                           int ldc) {
    // printf("--- 进入 cublas_matmul_wrapper ---\n");
    // printf("模板类型 InputType: %s\n", std::is_same_v<InputType, float> ?
    // "float"
    //                                    : std::is_same_v<InputType, hip_bfloat16>
    //                                        ? "hip_bfloat16"
    //                                        : "未知");

    // // 打印输入参数
    // printf("cuBLAS Handle: %p\n", (void *)handle);
    // printf("transa: %d (N=%d, T=%d)\n", transa, HIPBLAS_OP_N, HIPBLAS_OP_T);
    // printf("transb: %d (N=%d, T=%d)\n", transb, HIPBLAS_OP_N, HIPBLAS_OP_T);
    // printf("m: %d, n: %d, k: %d\n", m, n, k);
    // printf("alpha: %f (来自地址 %p)\n", *alpha, (void *)alpha);
    // printf("d_A: %p\n", (void *)d_A);
    // printf("lda: %d\n", lda);
    // printf("d_B: %p\n", (void *)d_B);
    // printf("ldb: %d\n", ldb);
    // printf("beta: %f (来自地址 %p)\n", *beta, (void *)beta);
    // printf("d_C: %p\n", (void *)d_C);
    // printf("ldc: %d\n", ldc);
    // fflush(stdout);  // 确保在调用 cuBLAS 前打印出来

    hipDataType cuda_data_type_A;
    hipDataType cuda_data_type_B;
    hipDataType cuda_data_type_C;

    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F_FAST_TF32;
    // printf("启用 TF32 计算类型\n");

    // hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;
    // printf("使用 FP32 计算类型\n");

    // 根据模板类型 InputType 确定 A, B, C 的 CUDA 数据类型
    if constexpr (std::is_same_v<InputType, hip_bfloat16>) {
        cuda_data_type_A = HIP_R_16BF;
        cuda_data_type_B = HIP_R_16BF;
        cuda_data_type_C = HIP_R_16BF;
        // printf("数据类型: A=BF16, B=BF16, C=BF16\n");
    } else if constexpr (std::is_same_v<InputType, float>) {
        cuda_data_type_A = HIP_R_32F;
        cuda_data_type_B = HIP_R_32F;
        cuda_data_type_C = HIP_R_32F;
        // printf("数据类型: A=FP32, B=FP32, C=FP32\n");
        // 可选: Ampere+ 可考虑 TF32 计算
        // compute_type = HIPBLAS_COMPUTE_32F_FAST_TF32;
    } else {
        // 这个 static_assert 会在编译时检查，如果运行到这里说明模板参数类型不对
        // 但为了运行时更明确，可以加个错误打印
        // fprintf(stderr, "错误：不支持的 InputType!\n");
        static_assert(std::is_same_v<InputType, hip_bfloat16> || std::is_same_v<InputType, float>,
                      "cublas_matmul_wrapper 只支持 hip_bfloat16 和 float "
                      "输入/输出类型。");
        return;  // 或者抛出异常
    }

    // printf("计算类型 compute_type: %d (HIPBLAS_COMPUTE_32F=%d)\n", compute_type,
    //        HIPBLAS_COMPUTE_32F);

    // // --- 选择算法 ---
    hipblasGemmAlgo_t algo = HIPBLAS_GEMM_DEFAULT;  // 让 cuBLAS 选择
    // printf("选择算法: HIPBLAS_GEMM_DEFAULT (%d)\n", algo);
    // fflush(stdout);  // 再次确保打印

    // // --- 执行 GEMM 操作 ---
    // printf("即将调用 hipblasGemmEx...\n");
    // fflush(stdout);

    hipblasStatus_t status = hipblasGemmEx(handle, transa, transb, m, n, k,
                                         alpha,             // 标量 alpha (主机)
                                         d_A,               // 矩阵 A (设备)
                                         cuda_data_type_A,  // A 的类型
                                         lda,
                                         d_B,               // 矩阵 B (设备)
                                         cuda_data_type_B,  // B 的类型
                                         ldb,
                                         beta,              // 标量 beta (主机)
                                         d_C,               // 矩阵 C (设备) - 类型现在是 InputType*
                                         cuda_data_type_C,  // C 的类型现在根据 InputType 确定
                                         ldc,
                                         compute_type,  // 内部计算精度 (推荐保持 FP32)
                                         algo);

    // printf("hipblasGemmEx 调用返回，状态码: %d\n", status);
    // fflush(stdout);

    // 使用 CHECK_CUBLAS 宏来检查返回状态
    CHECK_CUBLAS(status);

    // 可选：添加 CUDA 同步和错误检查，确保 GEMM 内核执行完成且没有异步错误
    // hipError_t cuda_err = hipDeviceSynchronize();
    // if (cuda_err != hipSuccess) {
    //   fprintf(stderr, "CUDA error after hipblasGemmEx sync: %s\n",
    //   hipGetErrorString(cuda_err));
    // } else {
    //    printf("hipDeviceSynchronize 成功\n");
    // }
    // fflush(stdout);

    // printf("--- 退出 cublas_matmul_wrapper ---\n");
    // fflush(stdout);  // 确保退出信息也打印出来
}
// --------------------------------------------------
// --------------------------------------------------
template <typename T>
void matmul(const Tensor<T> &A, const Tensor<T> &B, Tensor<T> *C, hipStream_t stream, const Tensor<T> *bias,
            int use_) {
    // 如果选择使用 cublas 计算，则调用 cublas 的包装函数接口

    const std::vector<size_t> &A_shape = A.sizes();
    const std::vector<size_t> &B_shape = B.sizes();
    // std::cout << A_shape[0] << " " << A_shape[1] << " " << B_shape[0] << " " << B_shape[1] << std::endl;

    // A: [M, K], B: [N, K]（保证 A 的第二维与 B 的第二维一致）
    size_t M = A_shape[0];
    size_t K = A_shape[1];
    size_t N = B_shape[1];

    // === 特殊处理: M=1的GEMV情况 ===
    if (M == 1) {
        // printf("使用 GEMV 优化分支 (M=1)\n");

        // 使用优化的GEMV kernel
        constexpr int ROWS_PER_BLOCK = 4;   // 每个block处理4个输出元素
        dim3 blockDim(32, ROWS_PER_BLOCK);  // 32线程构成一个warp，4个warp处理4个输出
        dim3 gridDim((N + ROWS_PER_BLOCK - 1) / ROWS_PER_BLOCK, 1);

        if (bias != nullptr) {
            // 检查偏置形状
            const std::vector<size_t> &bias_shape = bias->sizes();
            if (bias_shape.size() != 1) {
                throw std::runtime_error("Bias must be a 1D tensor");
            }
            if (bias_shape[0] != N) {
                throw std::runtime_error("Bias size must match output column dimension");
            }

            // printf("使用带bias的GEMV kernel\n");
            // 根据K的大小选择不同的优化策略
            if (K % 4 == 0 && K >= 128) {
                // K较大且是4的倍数，使用向量化版本
                gemv_with_bias_vectorized_kernel<T>
                    <<<gridDim, blockDim, 0, stream>>>(A.data_ptr(), B.data_ptr(), bias->data_ptr(), C->data_ptr(),
                                                       static_cast<int>(M), static_cast<int>(K), static_cast<int>(N));
            } else {
                // 使用标准版本
                gemv_with_bias_kernel<T><<<gridDim, blockDim, 0, stream>>>(A.data_ptr(), B.data_ptr(), bias->data_ptr(),
                                                                           C->data_ptr(), static_cast<int>(M),
                                                                           static_cast<int>(K), static_cast<int>(N));
            }
        } else {
            // printf("使用无bias的GEMV kernel\n");
            // 根据K的大小选择不同的优化策略
            if (K % 4 == 0 && K >= 128) {
                // K较大且是4的倍数，使用向量化版本
                gemv_vectorized_kernel<T><<<gridDim, blockDim, 0, stream>>>(A.data_ptr(), B.data_ptr(), C->data_ptr(),
                                                                            static_cast<int>(M), static_cast<int>(K),
                                                                            static_cast<int>(N));
            } else {
                // 使用标准版本
                gemv_kernel<T><<<gridDim, blockDim, 0, stream>>>(A.data_ptr(), B.data_ptr(), C->data_ptr(),
                                                                 static_cast<int>(M), static_cast<int>(K),
                                                                 static_cast<int>(N));
            }
        }

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            throw std::runtime_error("CUDA GEMV kernel launch failed: " + std::string(hipGetErrorString(err)));
        }
        return;
    }

    if (bias == nullptr && use_ == 2) {
        use_ = 1;
    }

    if (use_ == 2) {
        cutlass::Status status = run_cutlass_gemm_raw_templated<T,                             // ElementA
                                                                T,                             // ElementB
                                                                T,                             // ElementOutput
                                                                cutlass::layout::RowMajor,     // LayoutA
                                                                cutlass::layout::ColumnMajor,  // LayoutB
                                                                cutlass::layout::RowMajor,     // LayoutOutput
                                                                float,                         // ElementAccumulator
                                                                float,                         // ElementComputeEpilogue
                                                                cutlass::arch::OpClassTensorOp>(
            M, N, K, A.data_ptr(), B.data_ptr(), bias->data_ptr(), C->data_ptr(), stream);
    } else if (use_ == 1) {
        // 注意：这是直接CUDA算子库的实现，与统一算子库不同，它使用自己的static cublas句柄
        // 这是一个独立的实现，可以直接通过Tensor的matmul函数调用

        static hipblasHandle_t handle = nullptr;
        // 使用静态标志和互斥锁确保线程安全的单次初始化
        static std::once_flag init_flag;
        static std::mutex handle_mutex;  // 保护对 handle 的并发使用 (如果需要)

        // 确保 hipblasCreate 只被调用一次，且线程安全
        std::call_once(init_flag, [&]() {
            std::lock_guard<std::mutex> lock(handle_mutex);  // 锁定以进行创建
            // printf("--- Initializing static cublasHandle ---\n"); // 调试信息
            hipblasStatus_t status = hipblasCreate(&handle);
            if (status != HIPBLAS_STATUS_SUCCESS) {
                fprintf(stderr, "FATAL ERROR: hipblasCreate failed in static init: %d\n", status);
                // 在这里可能需要做更健壮的错误处理，比如抛出异常或终止程序
                handle = nullptr;  // 确保句柄无效
            } else {
                // hipStream_t stream = nullptr; // 或获取一个全局流
                // hipblasSetStream(handle, stream);
                // printf("--- Static cublasHandle initialized: %p ---\n",
                // (void*)handle); // 调试信息

                // 注册程序退出时调用的清理函数
                std::atexit([]() {
                    std::lock_guard<std::mutex> lock(handle_mutex);  // 同样锁定以进行销毁
                    if (handle != nullptr) {
                        // printf("--- Destroying static cublasHandle: %p ---\n",
                        // (void*)handle); // 调试信息
                        hipblasDestroy(handle);
                        handle = nullptr;  // 标记为已销毁
                    }
                });
            }
        });

        // 检查初始化是否成功 (如果在 call_once 中处理失败)
        if (handle == nullptr) {
            fprintf(stderr, "Error: cuBLAS handle was not initialized correctly.\n");
            return;  // 或者抛出异常
        }
        // 原始数据（均按行主序存储）：
        // A: M×K, 每行有 K 个元素  -> lda = K
        // B: N×K, 每行有 K 个元素  -> ldb = K
        // C: M×N, 每行有 N 个元素  -> ldc = N
        int lda = K;  // A 每行有 K 个元素
        int ldb = K;  // B 每行有 K 个元素
        int ldc = N;  // C 每行有 N 个元素
        const float alpha = 1.0f;
        const float beta = 0.0f;
        {  // 引入作用域方便 lock_guard 管理 虽然实际上并不十分需要（本项目暂不支持多线程操作）
            std::lock_guard<std::mutex> lock(handle_mutex);
            // 将 cuBLAS 操作与传入的 stream 关联
            CHECK_CUBLAS(hipblasSetStream(handle, stream));  // 确保 handle 使用正确的流

            // 目标计算： C = A * B^T
            // 利用转换： C^T = B * A^T
            // GEMM 调用参数：
            //    m = N, n = M, k = K
            // 同时对 A 和 B 使用转置操作：
            cublas_matmul_wrapper<T>(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, int(N), int(M), int(K), &alpha,
                                     B.data_ptr(),  // 原始 B
                                     ldb,           // ldb = K
                                     A.data_ptr(),  // 原始 A
                                     lda,           // lda = K
                                     &beta,
                                     C->data_ptr(),  // 输出 C
                                     ldc);           // ldc = N
                                                     // hipStreamSynchronize(stream);
        }
        // std::cout << "cublas_matmul_wrapper<T> 调用成功" << std::endl;
        if (bias != nullptr) {
            dim3 blockDim(16, 16);
            // 计算网格大小，确保覆盖所有元素
            dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);
            add_bias_kernel<T><<<gridDim, blockDim, 0, stream>>>(C->data_ptr(), bias->data_ptr(), static_cast<int>(M),
                                                                 static_cast<int>(N), ldc);
        }
        // std::cout << "add_bias_kernel 调用成功" << std::endl;
        return;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    if (bias == nullptr) {
        // 使用无偏置版本的kernel
        matmul_kernel<T><<<numBlocks, threadsPerBlock, 0, stream>>>(A.data_ptr(), B.data_ptr(), C->data_ptr(), M, K, N);
    } else {
        // 检查偏置形状
        const std::vector<size_t> &bias_shape = bias->sizes();
        if (bias_shape.size() != 1) {
            throw std::runtime_error("Bias must be a 1D tensor");
        }
        if (bias_shape[0] != N) {
            throw std::runtime_error("Bias size must match output column dimension");
        }
        // 使用带偏置版本的kernel
        matmul_with_bias_kernel<T><<<numBlocks, threadsPerBlock, 0, stream>>>(A.data_ptr(), B.data_ptr(),
                                                                              bias->data_ptr(), C->data_ptr(), M, K, N);
        // Use code with caution.
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error("CUDA kernel launch failed: " + std::string(hipGetErrorString(err)));
    }

    return;
}

template void matmul<float>(const Tensor<float> &, const Tensor<float> &, Tensor<float> *, hipStream_t,
                            const Tensor<float> *, int);
template void matmul<__hip_bfloat16>(const Tensor<__hip_bfloat16> &, const Tensor<__hip_bfloat16> &,
                                    Tensor<__hip_bfloat16> *, hipStream_t, const Tensor<__hip_bfloat16> *, int);

}  // namespace cuda_OP
