#include "hip/hip_runtime.h"
#include "cudaOP.cuh"
#include <cmath>

namespace cuda_OP
{

    // Each warp calculates one output element of the MLP layer.
    // This kernel fuses two GEMVs (gate and up projections), a SiLU activation,
    // and an element-wise multiplication.
    template <typename T, int WARP_SIZE = 32>
    __global__ void gemv_mlp_fused_kernel(
        const T *hidden_states, // Input tensor [1, hidden_size]
        const T *merged_weight, // Merged gate & up weights [2 * intermediate_size, hidden_size]
        T *output,              // Output tensor [1, intermediate_size]
        int hidden_size,
        int intermediate_size)
    {
        // Each warp is responsible for one output element `i` in the intermediate tensor.
        const int i = blockIdx.x * blockDim.y + threadIdx.y;
        if (i >= intermediate_size)
        {
            return;
        }

        const int lane = threadIdx.x;

        // Pointers to the start of the rows for the current output element `i`.
        const T *gate_weight_row = merged_weight + i * hidden_size;
        const T *up_weight_row = merged_weight + (i + intermediate_size) * hidden_size;

        float gate_val_acc = 0.0f;
        float up_val_acc = 0.0f;

        // Use float4 for vectorized memory access.
        constexpr int VEC_UNIT = sizeof(float4) / sizeof(T);

        // Loop over the hidden_size dimension, with each thread handling a `VEC_UNIT` chunk.
        for (int k = lane * VEC_UNIT; k < hidden_size; k += WARP_SIZE * VEC_UNIT)
        {
            // Load vectors from hidden_states, gate_weight, and up_weight.
            Vec<T, VEC_UNIT> v_hidden, v_gate, v_up;
            v_hidden.f4 = *reinterpret_cast<const float4 *>(hidden_states + k);
            v_gate.f4 = *reinterpret_cast<const float4 *>(gate_weight_row + k);
            v_up.f4 = *reinterpret_cast<const float4 *>(up_weight_row + k);

// Perform dot product for both projections.
#pragma unroll
            for (int j = 0; j < VEC_UNIT; ++j)
            {
                gate_val_acc += static_cast<float>(v_hidden.t[j]) * static_cast<float>(v_gate.t[j]);
                up_val_acc += static_cast<float>(v_hidden.t[j]) * static_cast<float>(v_up.t[j]);
            }
        }

        // Reduce the partial sums within the warp.
        for (int offset = WARP_SIZE / 2; offset > 0; offset >>= 1)
        {
            gate_val_acc += __shfl_xor_sync(0xffffffff, gate_val_acc, offset);
            up_val_acc += __shfl_xor_sync(0xffffffff, up_val_acc, offset);
        }
        if (lane == 0)
        {

            float silu_val = gate_val_acc * (1.0f / (1.0f + expf(-gate_val_acc)));

            // Element-wise multiplication
            float final_val = silu_val * up_val_acc;

            output[i] = static_cast<T>(final_val);
        }
    }

    template <typename T>
    void gemv_mlp_fused(
        const Tensor<T> *hidden_states,
        const Tensor<T> *merged_mlp_weight,
        Tensor<T> *output,
        hipStream_t stream)
    {
        const int hidden_size = hidden_states->sizes()[1];

        const int intermediate_size = merged_mlp_weight->sizes()[1] / 2;
        const T *d_hidden = hidden_states->data_ptr();
        const T *d_merged_w = merged_mlp_weight->data_ptr();
        T *d_output = output->data_ptr();

        // Standard launch configuration for GEMV-like kernels.
        constexpr int ROWS_PER_BLOCK = 4;
        dim3 blockDim(32, ROWS_PER_BLOCK);
        dim3 gridDim((intermediate_size + ROWS_PER_BLOCK - 1) / ROWS_PER_BLOCK, 1);

        gemv_mlp_fused_kernel<T><<<gridDim, blockDim, 0, stream>>>(
            d_hidden,
            d_merged_w,
            d_output,
            hidden_size,
            intermediate_size);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            throw std::runtime_error("gemv_mlp_fused_kernel launch failed: " + std::string(hipGetErrorString(err)));
        }
    }

    // Explicit template instantiations
    template void gemv_mlp_fused<hip_bfloat16>(
        const Tensor<hip_bfloat16> *hidden_states,
        const Tensor<hip_bfloat16> *merged_mlp_weight,
        Tensor<hip_bfloat16> *output,
        hipStream_t stream);

    template void gemv_mlp_fused<float>(
        const Tensor<float> *hidden_states,
        const Tensor<float> *merged_mlp_weight,
        Tensor<float> *output,
        hipStream_t stream);

} // namespace cuda_OP
