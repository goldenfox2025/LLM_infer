#include "hip/hip_runtime.h"
#ifndef CUDA_OP_CUH
#define CUDA_OP_CUH

#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>

#include <limits>
#include <stdexcept>
#include <string>  // 用于 std::to_string
#include <vector>

// 包含 CUB 头文件
#include <hipcub/hipcub.hpp>

// 假设这些是项目特定的头文件 (忽略其具体内容)
#include "CudaMemoryPool.hpp"
#include "cudaOP.cuh"
#include "tensor.hpp"

// --- 常量定义 (cudaOP.cuh定义) ---
// 示例: 定义 Kernel 2 中共享内存数组的最大大小
// #define MAX_TOPK 256 // 重要: 必须定义此宏, 否则 Kernel 2 无法编译!
// 假设 MAX_TOPK 在其他地方定义
#define MAX_TOPK 1024
// --- 检查 CUDA 错误的宏 (cudaOP.cuh定义) ---
// #define CUDA_CHECK(call) ... // 重要: 必须定义此宏!
// 假设 CUDA_CHECK 在其他地方定义

namespace cuda_OP {

// Kernel 1 (融合): 缩放 Logits 并初始化索引 (多块执行)
// 功能: 将输入的 logits 除以 temperature 并初始化一个从 0 到 vocab_size-1
// 的索引数组。
template <typename T>
__global__ void scale_logits_and_init_indices_kernel(const T* __restrict__ logits,  // 输入: 原始 logits (设备指针)
                                                     T* d_scaled_logits,            // 输出: 缩放后的 logits (设备指针)
                                                     int* d_indices,                // 输出: 初始化的索引数组 (设备指针)
                                                     size_t vocab_size,             // 输入: 词汇表大小
                                                     float temperature              // 输入: 温度系数
) {
    // 使用网格跨步循环处理所有词汇
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < vocab_size; i += stride) {
        // 1. 缩放 Logits
        float logit_f;
        // 使用 __ldg 进行缓存的全局内存读取
        if constexpr (std::is_same_v<T, __hip_bfloat16>) {
            logit_f = __bfloat162float(__ldg(&logits[i]));
        } else {
            logit_f = static_cast<float>(__ldg(&logits[i]));
        }
        float scaled_logit_f = logit_f / temperature;  // 应用温度缩放

        // 写回缩放后的值 (根据类型转换)
        if constexpr (std::is_same_v<T, __hip_bfloat16>) {
            d_scaled_logits[i] = __float2bfloat16(scaled_logit_f);
        } else {
            d_scaled_logits[i] = static_cast<T>(scaled_logit_f);
        }

        // 2. 初始化索引
        d_indices[i] = i;
    }
}

// Kernel 2: 从 Top-K 结果中进行最终采样 (单块执行)
// 功能: 对排序后的 Top-K logits 进行 softmax 和加权随机采样。
// T: 数据类型 (float 或 __hip_bfloat16)
// BLOCK_DIM_X: CUDA 块的大小 (用于 CUB 和并行计算)
template <typename T, int BLOCK_DIM_X>
__global__ void sample_from_sorted_topk_kernel(const T* __restrict__ d_sorted_topk_logits,     // 输入: 排序后的 Top-K
                                                                                               // logits (设备指针)
                                               const int* __restrict__ d_sorted_topk_indices,  // 输入: 排序后的 Top-K
                                                                                               // 索引 (设备指针)
                                               size_t k,  // 输入: Top-K 中的 'k' 值 (必须 <= MAX_TOPK)
                                               const float* __restrict__ d_max_val_ptr,  // 输入: 所有缩放后 logits
                                                                                         // 的最大值 (设备指针)
                                               hiprandState* states,       // 输入/输出: cuRAND 状态 (用于随机数生成)
                                               uint32_t* d_sampled_index  // 输出: 最终采样得到的索引 (设备指针)
) {
    // CUB 块内归约，用于计算 exp 值的总和
    using BlockReduce = hipcub::BlockReduce<float, BLOCK_DIM_X>;

    // 共享内存: 用于 CUB 临时存储和存储 Top-K 的 exp(logit - max_val) 值
    __shared__ union SharedStorage {
        typename BlockReduce::TempStorage reduce_storage;  // CUB Reduce 所需的存储
        // 联合体确保内存复用。需要足够空间存放 MAX_TOPK 个 float 值。
        struct Combined {
            typename BlockReduce::TempStorage reduce_storage;
            float exp_vals[MAX_TOPK];  // 存储 Top-K 指数的数组 (大小由 MAX_TOPK 决定)
        } combined;
    } shared_storage;

    int tid = threadIdx.x;  // 当前线程 ID

    // 线程 0 读取最大值并存入共享内存
    __shared__ float max_val_shared;
    if (tid == 0) {
        max_val_shared = *d_max_val_ptr;
    }
    __syncthreads();  // 确保所有线程都能读到 max_val_shared

    // --- 并行计算 exp(logit - max_val) ---
    float thread_exp_sum = 0.0f;  // 每个线程的局部 exp 值累加和
    // 线程协作计算前 k 个值的 exp
    for (int i = tid; i < k; i += BLOCK_DIM_X) {
        T scaled_logit_T = d_sorted_topk_logits[i];
        float scaled_logit_f;
        // 类型转换
        if constexpr (std::is_same_v<T, __hip_bfloat16>) {
            scaled_logit_f = __bfloat162float(scaled_logit_T);
        } else {
            scaled_logit_f = static_cast<float>(scaled_logit_T);
        }
        // 计算 exp(logit - max)，减去 max 防止上溢
        float exp_val = expf(scaled_logit_f - max_val_shared);

        // 将计算出的 exp 值存入共享内存，供后续采样使用
        if (i < MAX_TOPK) {  // 检查边界，确保不越界写入共享数组
            shared_storage.combined.exp_vals[i] = exp_val;
        }
        // 累加到线程局部和
        thread_exp_sum += exp_val;
    }
    __syncthreads();  // 确保所有 exp_vals
                      // 都已写入共享内存，并且所有线程都完成了计算

    // --- 使用 CUB 在块内归约求和 ---
    // CUB 调用: BlockReduce::Sum
    // 输入: thread_exp_sum (每个线程计算的部分 exp 和)
    // 输出: block_total_exp_sum (块内所有线程的 exp 总和)
    // 作用: 高效地计算块内所有线程的部分和的总和。
    // 临时存储: shared_storage.combined.reduce_storage (在共享内存中)
    float block_total_exp_sum = BlockReduce(shared_storage.combined.reduce_storage).Sum(thread_exp_sum);
    // 此刻，块内所有线程的 block_total_exp_sum 都持有相同的总和值

    // --- 线程 0 执行加权采样 ---
    if (tid == 0) {
        float total_exp_sum = block_total_exp_sum;  // 获取总和
        hiprandState localState = states[0];         // 获取 cuRAND 状态

        uint32_t selected_final_index = 0;  // 初始化采样结果

        // 处理特殊情况：如果总和过小或 k=0，则默认选择第一个 top-k 元素 (或 0)
        if (total_exp_sum <= 1e-9f || k == 0) {
            if (k > 0) {  // 如果 k>0 但总和接近 0，选择概率最高的那个
                selected_final_index = static_cast<uint32_t>(d_sorted_topk_indices[0]);
            } else {  // 如果 k=0 (理论上不应发生，因为前面有检查)，返回 0
                selected_final_index = 0;
            }
        } else {
            // 生成一个 [0, total_exp_sum) 范围内的随机数
            float r = hiprand_uniform(&localState) * total_exp_sum;
            float cumulative = 0.0f;  // 累积概率

            // 线性扫描共享内存中的 exp 值进行加权采样
            selected_final_index = static_cast<uint32_t>(d_sorted_topk_indices[0]);  // 默认值
            float* s_exp_vals = shared_storage.combined.exp_vals;                    // 指向共享内存中的 exp 数组
            for (int i = 0; i < k; ++i) {
                // 从共享内存读取预先计算好的 exp 值
                cumulative += s_exp_vals[i];
                // 如果累积和超过随机阈值 r，则选择当前索引
                if (cumulative >= r) {
                    selected_final_index = static_cast<uint32_t>(d_sorted_topk_indices[i]);
                    break;  // 找到后即退出循环
                }
            }
        }
        // 将最终选定的索引写入输出指针
        *d_sampled_index = selected_final_index;
        // 更新 cuRAND 状态
        states[0] = localState;
    }
}

// CUB TransformIterator 的辅助 Functor
// 功能: 在 CUB 操作中动态地将输入类型 Tin (如 bfloat16) 转换为 float。
template <typename Tin>
struct ConvertToFloatFunctor {
    __device__ __forceinline__ float operator()(const Tin& x) const {
        if constexpr (std::is_same_v<Tin, __hip_bfloat16>) {
            return __bfloat162float(x);  // bfloat16 转 float
        } else {
            return static_cast<float>(x);  // 其他类型直接转 float
        }
    }
};

// 主采样函数
// 功能: 对输入的 logits 执行 Top-K 采样。
// 输入:
//   - logits: 输入的 logits 张量 (T 类型, 形状 [seq_len, vocab_size], 必须在
//   CUDA 设备上)
//   - temperature: 温度系数，用于缩放 logits
//   - top_p: Top-P 采样的概率阈值 (当前代码中未使用)
//   - top_k: Top-K 采样的 K 值
//   - d_states: 指向设备端 cuRAND 状态的指针
//   - stream: CUDA 流
// 返回:
//   - 指向设备端存储最终采样结果 (单个 uint32_t 索引) 的指针。注意：结果仍在
//   GPU 上。
template <typename T>
uint32_t* sample(Tensor<T>&& logits, float temperature,
                 float top_p,  // top_p 未在此实现中使用
                 size_t top_k, hiprandState* d_states, hipStream_t stream) {
    // --- 输入验证 ---
    if (logits.device() != Device::CUDA) {
        throw std::runtime_error("输入张量必须在 CUDA 设备上");
    }
    // Top-K 采样至少需要 k=1
    if (top_k == 0) {
        throw std::runtime_error("top_k 必须至少为 1");
    }

    const auto& shape = logits.sizes();
    if (shape.size() != 2 || shape[0] == 0 || shape[1] == 0) {
        throw std::runtime_error("输入张量必须是二维且维度非零 [seq_len, vocab_size]");
    }

    const size_t seq_len = shape[0];
    const size_t vocab_size = shape[1];

    // 如果 K 大于词汇表大小，则将其限制为词汇表大小
    if (top_k > vocab_size) {
        top_k = vocab_size;
    }
    // 检查 K 是否超过 Kernel 2 中共享内存的限制 (MAX_TOPK)
    if (top_k > MAX_TOPK) {
        throw std::runtime_error("请求的 top_k (" + std::to_string(top_k) + ") 超过了 Kernel 2 的 MAX_TOPK 限制 (" +
                                 std::to_string(MAX_TOPK) + ")，无法在共享内存中分配。");
    }

    // 获取指向最后一个 token 的 logits 数据的设备指针
    const T* d_logits_ptr = logits.data_ptr() + (seq_len - 1) * vocab_size;

    // --- 内存管理 ---
    auto& pool = GlobalCudaMemoryPool::instance();  // 获取全局内存池实例
    // 分配临时设备内存
    T* d_scaled_logits = static_cast<T*>(pool.allocate(vocab_size * sizeof(T)));         // 存储缩放后的 logits
    float* d_max_val = static_cast<float*>(pool.allocate(sizeof(float)));                // 存储缩放后 logits 的最大值
    int* d_indices = static_cast<int*>(pool.allocate(vocab_size * sizeof(int)));         // 存储原始索引
    T* d_sorted_logits = static_cast<T*>(pool.allocate(vocab_size * sizeof(T)));         // 存储排序后的 logits
    int* d_sorted_indices = static_cast<int*>(pool.allocate(vocab_size * sizeof(int)));  // 存储排序后的索引
    // 使用tagged memory分配采样结果，确保每次都是相同的固定地址
    uint32_t* d_sampled_index = static_cast<uint32_t*>(pool.allocate(sizeof(uint32_t), "sample_output"));

    // CUB 临时存储指针和大小 (初始为 nullptr 和 0)
    void* d_reduce_temp_storage = nullptr;
    size_t reduce_temp_storage_bytes = 0;
    void* d_sort_temp_storage = nullptr;
    size_t sort_temp_storage_bytes = 0;

    // --- 步骤 1 (融合 Kernel): 缩放 Logits 并初始化索引 ---
    const int scale_init_block_size = 256;  // 定义块大小
    const int scale_init_grid_size =        // 计算网格大小
        (vocab_size + scale_init_block_size - 1) / scale_init_block_size;
    // 启动 Kernel 1
    scale_logits_and_init_indices_kernel<T><<<scale_init_grid_size, scale_init_block_size, 0, stream>>>(
        d_logits_ptr, d_scaled_logits, d_indices, vocab_size, temperature);
    CUDA_CHECK(hipGetLastError());  // 检查核函数启动错误

    // --- 步骤 2: 查找最大缩放 Logit (使用 CUB Device Reduce) ---
    // 创建一个转换迭代器，在计算 Max 时将 T 动态转换为 float
    hipcub::TransformInputIterator<float, ConvertToFloatFunctor<T>, const T*> itr(d_scaled_logits,
                                                                               ConvertToFloatFunctor<T>());
    // 第一次调用 CUB Reduce: 获取所需的临时存储大小
    // CUB 调用: DeviceReduce::Max (第一次调用)
    // 输入: itr (转换迭代器), vocab_size (元素数量)
    // 输出: reduce_temp_storage_bytes (所需的临时存储大小)
    // 作用: 计算执行 Max 操作所需的临时设备内存大小。
    CUDA_CHECK(
        hipcub::DeviceReduce::Max(d_reduce_temp_storage, reduce_temp_storage_bytes, itr, d_max_val, vocab_size, stream));
    // 分配所需的临时存储
    d_reduce_temp_storage = pool.allocate(reduce_temp_storage_bytes);
    // 第二次调用 CUB Reduce: 执行 Max 操作
    // CUB 调用: DeviceReduce::Max (第二次调用)
    // 输入: itr (转换迭代器), vocab_size, d_reduce_temp_storage (临时存储指针)
    // 输出: d_max_val (指向最大值的设备指针)
    // 作用: 在设备上计算输入范围内的最大值。
    CUDA_CHECK(
        hipcub::DeviceReduce::Max(d_reduce_temp_storage, reduce_temp_storage_bytes, itr, d_max_val, vocab_size, stream));
    CUDA_CHECK(hipGetLastError());  // 检查 CUB 调用错误

    // --- 步骤 3: 按 Logit 值降序排序 (Logit, Index) 对 (使用 CUB Device Radix
    // Sort) --- 第一次调用 CUB Sort: 获取所需的临时存储大小 CUB 调用:
    // DeviceRadixSort::SortPairsDescending (第一次调用) 输入: d_scaled_logits
    // (键), d_indices (值), vocab_size 输出: sort_temp_storage_bytes
    // (所需的临时存储大小) 作用: 计算执行降序键值对排序所需的临时设备内存大小。
    CUDA_CHECK(hipcub::DeviceRadixSort::SortPairsDescending(d_sort_temp_storage, sort_temp_storage_bytes, d_scaled_logits,
                                                         d_sorted_logits, d_indices, d_sorted_indices, vocab_size, 0,
                                                         sizeof(T) * 8, stream));  // sizeof(T)*8 是 T 类型的位数
    // 分配所需的临时存储
    d_sort_temp_storage = pool.allocate(sort_temp_storage_bytes);
    // 第二次调用 CUB Sort: 执行排序操作
    // CUB 调用: DeviceRadixSort::SortPairsDescending (第二次调用)
    // 输入: d_scaled_logits, d_indices, vocab_size, d_sort_temp_storage
    // (临时存储指针) 输出: d_sorted_logits (排序后的键), d_sorted_indices
    // (排序后的值) 作用: 根据键 (logits) 对键值对 (logit, index) 进行降序排序。
    CUDA_CHECK(hipcub::DeviceRadixSort::SortPairsDescending(d_sort_temp_storage, sort_temp_storage_bytes, d_scaled_logits,
                                                         d_sorted_logits, d_indices, d_sorted_indices, vocab_size, 0,
                                                         sizeof(T) * 8, stream));
    CUDA_CHECK(hipGetLastError());  // 检查 CUB 调用错误

    // --- 步骤 4: 从 Top-K 结果中进行最终加权采样 (单块 Kernel) ---
    // 为采样核函数选择块大小 (必须与核函数模板参数匹配)
    const int sample_block_size = 128;  // 示例块大小

    // 计算 Kernel 2 所需的共享内存大小
    // 需要 CUB Reduce 的临时存储大小 + 存储 exp 值的数组大小
    // 注意: 精确计算 CUB Reduce 存储需要模板特化，这里用 sizeof 估算
    size_t reduce_storage_size_est = sizeof(hipcub::BlockReduce<float,
                                                             sample_block_size>::TempStorage);  // CUB Reduce 存储
    size_t exp_values_size = MAX_TOPK * sizeof(float);                     // 存储 exp 值的数组大小 (基于 MAX_TOPK)
    size_t sample_shared_mem = reduce_storage_size_est + exp_values_size;  // 总共享内存需求

    // 检查所需共享内存是否超过设备限制
    int max_shared_mem_per_block = 0;
    CUDA_CHECK(hipDeviceGetAttribute(&max_shared_mem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
    if (sample_shared_mem > max_shared_mem_per_block) {
        throw std::runtime_error("计算出的所需共享内存 (" + std::to_string(sample_shared_mem) + ") 超过设备限制 (" +
                                 std::to_string(max_shared_mem_per_block) +
                                 ")。检查 MAX_TOPK 或减小 sample_block_size。");
    }

    // 启动 Kernel 2 (单块，块大小为 sample_block_size)
    // 模板参数 <T, sample_block_size> 必须与核函数定义匹配
    sample_from_sorted_topk_kernel<T, sample_block_size><<<1, sample_block_size, sample_shared_mem, stream>>>(
        d_sorted_logits,   // 排序后的 top-k logits
        d_sorted_indices,  // 排序后的 top-k indices
        top_k,             // 实际使用的 k 值 (已确保 <= MAX_TOPK)
        d_max_val,         // 最大 logit 值指针
        d_states,          // cuRAND 状态
        d_sampled_index    // 输出采样的索引
    );
    CUDA_CHECK(hipGetLastError());  // 检查核函数启动错误

    // --- 结果处理 ---
    // 注意: 此函数返回的是指向设备内存中结果的指针 `d_sampled_index`。
    // 如果需要将结果复制回主机，需要在此处添加 hipMemcpy 调用。
    // uint32_t h_result = 0;
    // CUDA_CHECK(hipMemcpy(&h_result, d_sampled_index, sizeof(uint32_t),
    // hipMemcpyDeviceToHost));

    // --- 释放临时内存 ---
    // 将所有临时分配的设备内存返还给内存池
    pool.free(d_scaled_logits);
    pool.free(d_max_val);
    pool.free(d_indices);
    pool.free(d_sorted_logits);
    pool.free(d_sorted_indices);
    // pool.free(d_sampled_index); // 不释放，因为它是返回值
    pool.free(d_reduce_temp_storage);
    pool.free(d_sort_temp_storage);

    // 返回指向设备端采样结果的指针
    return d_sampled_index;
}
#include "common.hpp"
// 批量采样函数
// 功能: 对输入的每个序列位置进行采样，返回指向设备端采样结果的指针数组
// 输入:
//   - logits: 输入的 logits 张量 (T 类型, 形状 [seq_len, vocab_size], 必须在
//   CUDA 设备上)
//   - temperature: 温度系数，用于缩放 logits
//   - top_p: Top-P 采样的概率阈值 (当前代码中未使用)
//   - top_k: Top-K 采样的 K 值
//   - d_states: 指向设备端 cuRAND 状态的指针
//   - stream: CUDA 流
// 返回:
//   - 包含指向设备端存储采样结果的指针数组，每个指针对应一个序列位置的采样结果
template <typename T>
std::vector<uint32_t*> sample_batch(Tensor<T>&& logits, float temperature,
                                    float top_p,  // top_p 未在此实现中使用
                                    size_t top_k, hiprandState* d_states, hipStream_t stream) {
    // --- 输入验证 ---
    if (logits.device() != Device::CUDA) {
        throw std::runtime_error("输入张量必须在 CUDA 设备上");
    }
    // Top-K 采样至少需要 k=1
    if (top_k == 0) {
        throw std::runtime_error("top_k 必须至少为 1");
    }

    const auto& shape = logits.sizes();
    if (shape.size() != 2 || shape[0] == 0 || shape[1] == 0) {
        throw std::runtime_error("输入张量必须是二维且维度非零 [seq_len, vocab_size]");
    }

    const size_t seq_len = shape[0];
    const size_t vocab_size = shape[1];

    // 如果 K 大于词汇表大小，则将其限制为词汇表大小
    if (top_k > vocab_size) {
        top_k = vocab_size;
    }
    // 检查 K 是否超过 Kernel 2 中共享内存的限制 (MAX_TOPK)
    if (top_k > MAX_TOPK) {
        throw std::runtime_error("请求的 top_k (" + std::to_string(top_k) + ") 超过了 Kernel 2 的 MAX_TOPK 限制 (" +
                                 std::to_string(MAX_TOPK) + ")，无法在共享内存中分配。");
    }

    // 创建结果向量，存储每个序列位置的采样结果指针
    std::vector<uint32_t*> result_tokens;
    result_tokens.reserve(seq_len);

    // 为每个序列位置创建一个视图并调用sample函数
    for (size_t i = 0; i < seq_len; i++) {
        // 创建当前位置logits的视图
        // 使用slice方法，需要为每个维度提供start和end向量
        std::vector<size_t> start = {i, 0};
        std::vector<size_t> end = {i + 1, vocab_size};
        Tensor<T> logit_view = logits.slice(start, end);
        uint32_t* token_ptr = sample(std::move(logit_view), temperature, top_p, top_k, d_states, stream);
        result_tokens.push_back(token_ptr);
    }

    return result_tokens;
}

// 采样函数的变体，将结果写入指定的GPU内存位置
template <typename T>
void sample_to_fixed(Tensor<T>&& input, uint32_t* output_ptr, float temperature, float top_p, size_t top_k,
                     hiprandState* d_states, hipStream_t stream) {
    // 使用原始sample函数获取采样结果
    uint32_t* sampled_ptr = sample(std::move(input), temperature, top_p, top_k, d_states, stream);

    // 将结果复制到指定的输出位置
    hipMemcpyAsync(output_ptr, sampled_ptr, sizeof(uint32_t), hipMemcpyDeviceToDevice, stream);

    // 释放原始sample函数分配的内存
    hipFree(sampled_ptr);
}

// 批量采样函数的变体，将结果写入指定的GPU内存位置数组
template <typename T>
void sample_batch_to_fixed(Tensor<T>&& logits, uint32_t* output_ptr, float temperature, float top_p, size_t top_k,
                           hiprandState* d_states, hipStream_t stream) {
    // --- 输入验证 ---
    if (logits.device() != Device::CUDA) {
        throw std::runtime_error("输入张量必须在 CUDA 设备上");
    }
    // Top-K 采样至少需要 k=1
    if (top_k == 0) {
        throw std::runtime_error("top_k 必须至少为 1");
    }

    const auto& shape = logits.sizes();
    if (shape.size() != 2 || shape[0] == 0 || shape[1] == 0) {
        throw std::runtime_error("输入张量必须是二维且维度非零 [seq_len, vocab_size]");
    }

    const size_t seq_len = shape[0];
    const size_t vocab_size = shape[1];

    // 为每个序列位置创建一个视图并调用sample_to_fixed函数
    for (size_t i = 0; i < seq_len; i++) {
        // 创建当前位置logits的视图
        std::vector<size_t> start = {i, 0};
        std::vector<size_t> end = {i + 1, vocab_size};
        Tensor<T> logit_view = logits.slice(start, end);

        // 调用单个token的sample_to_fixed函数，将结果写入output_ptr[i]
        sample_to_fixed(std::move(logit_view), output_ptr + i, temperature, top_p, top_k, d_states, stream);
    }
}

// --- 模板显式实例化 ---
// 为 float 和 __hip_bfloat16 类型实例化 sample 函数
template uint32_t* sample<float>(Tensor<float>&&, float, float, size_t, hiprandState*, hipStream_t);
template uint32_t* sample<__hip_bfloat16>(Tensor<__hip_bfloat16>&&, float, float, size_t, hiprandState*, hipStream_t);

// 为 float 和 __hip_bfloat16 类型实例化 sample_to_fixed 函数
template void sample_to_fixed<float>(Tensor<float>&&, uint32_t*, float, float, size_t, hiprandState*, hipStream_t);
template void sample_to_fixed<__hip_bfloat16>(Tensor<__hip_bfloat16>&&, uint32_t*, float, float, size_t, hiprandState*,
                                             hipStream_t);

// 为 float 和 __hip_bfloat16 类型实例化 sample_batch 函数
template std::vector<uint32_t*> sample_batch<float>(Tensor<float>&&, float, float, size_t, hiprandState*, hipStream_t);
template std::vector<uint32_t*> sample_batch<__hip_bfloat16>(Tensor<__hip_bfloat16>&&, float, float, size_t, hiprandState*,
                                                            hipStream_t);

// 为 float 和 __hip_bfloat16 类型实例化 sample_batch_to_fixed 函数
template void sample_batch_to_fixed<float>(Tensor<float>&&, uint32_t*, float, float, size_t, hiprandState*,
                                           hipStream_t);
template void sample_batch_to_fixed<__hip_bfloat16>(Tensor<__hip_bfloat16>&&, uint32_t*, float, float, size_t,
                                                   hiprandState*, hipStream_t);

}  // namespace cuda_OP

#endif  // CUDA_OP_CUH