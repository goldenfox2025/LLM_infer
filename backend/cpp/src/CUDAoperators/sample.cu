#include "hip/hip_runtime.h"
#ifndef CUDA_OP_CUH
#define CUDA_OP_CUH

#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>

#include <limits>
#include <stdexcept>
#include <string>  // 用于 std::to_string
#include <vector>

// 包含 CUB 头文件
#include <hipcub/hipcub.hpp>

// 假设这些是项目特定的头文件 (忽略其具体内容)
#include "CudaMemoryPool.hpp"
#include "cudaOP.cuh"
#include "tensor.hpp"

// --- 常量定义 (cudaOP.cuh定义) ---
// 示例: 定义 Kernel 2 中共享内存数组的最大大小
// #define MAX_TOPK 256 // 重要: 必须定义此宏, 否则 Kernel 2 无法编译!
// 假设 MAX_TOPK 在其他地方定义
#define MAX_TOPK 1024
// --- 检查 CUDA 错误的宏 (cudaOP.cuh定义) ---
// #define CUDA_CHECK(call) ... // 重要: 必须定义此宏!
// 假设 CUDA_CHECK 在其他地方定义

namespace cuda_OP {

// Kernel 1 (融合): 缩放 Logits 并初始化索引 (多块执行)
// 功能: 将输入的 logits 除以 temperature 并初始化一个从 0 到 vocab_size-1
// 的索引数组。
template <typename T>
__global__ void scale_logits_and_init_indices_kernel(const T* __restrict__ logits,  // 输入: 原始 logits (设备指针)
                                                     T* d_scaled_logits,            // 输出: 缩放后的 logits (设备指针)
                                                     int* d_indices,                // 输出: 初始化的索引数组 (设备指针)
                                                     size_t vocab_size,             // 输入: 词汇表大小
                                                     float temperature              // 输入: 温度系数
) {
    // 使用网格跨步循环处理所有词汇
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < vocab_size; i += stride) {
        // 1. 缩放 Logits
        float logit_f;
        // 使用 __ldg 进行缓存的全局内存读取
        if constexpr (std::is_same_v<T, __hip_bfloat16>) {
            logit_f = __bfloat162float(__ldg(&logits[i]));
        } else {
            logit_f = static_cast<float>(__ldg(&logits[i]));
        }
        float scaled_logit_f = logit_f / temperature;  // 应用温度缩放

        // 写回缩放后的值 (根据类型转换)
        if constexpr (std::is_same_v<T, __hip_bfloat16>) {
            d_scaled_logits[i] = __float2bfloat16(scaled_logit_f);
        } else {
            d_scaled_logits[i] = static_cast<T>(scaled_logit_f);
        }

        // 2. 初始化索引
        d_indices[i] = i;
    }
}

// 将 logits 缩放为 float 并初始化索引
template <typename T>
__global__ void scale_logits_to_float_and_init_indices_kernel(const T* __restrict__ logits,
                                                              float* __restrict__ scaled_logits_f,
                                                              int* __restrict__ indices, size_t vocab_size,
                                                              float temperature) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < static_cast<int>(vocab_size); i += stride) {
        float v = 0.0f;
        if constexpr (std::is_same_v<T, __hip_bfloat16>) {
            v = __bfloat162float(__ldg(&logits[i]));
        } else {
            v = static_cast<float>(__ldg(&logits[i]));
        }
        scaled_logits_f[i] = v / temperature;
        indices[i] = i;
    }
}

// Kernel 2: 从 Top-K 结果中进行最终采样 (单块执行)
// 功能: 对排序后的 Top-K logits 进行 softmax 和加权随机采样。
// BLOCK_DIM_X: CUDA 块的大小 (用于 CUB 和并行计算)
template <typename T, int BLOCK_DIM_X>
__global__ void sample_from_sorted_topk_kernel(
    const T* __restrict__ d_sorted_topk_logits,     // 输入: 排序后的 Top-K logits
    const int* __restrict__ d_sorted_topk_indices,  // 输入: 排序后的 Top-K 索引
    size_t k,                                       // Top-K
    const float* __restrict__ d_max_val_ptr,        // 最大值
    hiprandState* states,                            // cuRAND 状态
    uint32_t* d_sampled_index) {                    // 输出索引
    // CUB 块内归约，用于计算 exp 值的总和
    using BlockReduce = hipcub::BlockReduce<float, BLOCK_DIM_X>;

    // 共享内存: 用于 CUB 临时存储和存储 Top-K 的 exp(logit - max_val) 值
    __shared__ union SharedStorage {
        typename BlockReduce::TempStorage reduce_storage;  // CUB Reduce 所需的存储
        // 联合体确保内存复用。需要足够空间存放 MAX_TOPK 个 float 值。
        struct Combined {
            typename BlockReduce::TempStorage reduce_storage;
            float exp_vals[MAX_TOPK];  // 存储 Top-K 指数的数组 (大小由 MAX_TOPK 决定)
        } combined;
    } shared_storage;

    int tid = threadIdx.x;  // 当前线程 ID

    // 线程 0 读取最大值并存入共享内存
    __shared__ float max_val_shared;
    if (tid == 0) {
        max_val_shared = *d_max_val_ptr;
    }
    __syncthreads();  // 确保所有线程都能读到 max_val_shared

    // --- 并行计算 exp(logit - max_val) ---
    float thread_exp_sum = 0.0f;  // 每个线程的局部 exp 值累加和
    // 线程协作计算前 k 个值的 exp
    for (int i = tid; i < k; i += BLOCK_DIM_X) {
        float scaled_logit_f;
        if constexpr (std::is_same_v<T, __hip_bfloat16>) {
            scaled_logit_f = __bfloat162float(d_sorted_topk_logits[i]);
        } else {
            scaled_logit_f = static_cast<float>(d_sorted_topk_logits[i]);
        }
        // 计算 exp(logit - max)，减去 max 防止上溢
        float exp_val = expf(scaled_logit_f - max_val_shared);

        // 将计算出的 exp 值存入共享内存，供后续采样使用
        if (i < MAX_TOPK) {  // 检查边界，确保不越界写入共享数组
            shared_storage.combined.exp_vals[i] = exp_val;
        }
        // 累加到线程局部和
        thread_exp_sum += exp_val;
    }
    __syncthreads();  // 确保所有 exp_vals
                      // 都已写入共享内存，并且所有线程都完成了计算

    // --- 使用 CUB 在块内归约求和 ---
    float block_total_exp_sum = BlockReduce(shared_storage.combined.reduce_storage).Sum(thread_exp_sum);
    // 此刻，块内所有线程的 block_total_exp_sum 都持有相同的总和值

    // --- 线程 0 执行加权采样 ---
    if (tid == 0) {
        float total_exp_sum = block_total_exp_sum;  // 获取总和
        hiprandState localState = states[0];         // 获取 cuRAND 状态

        uint32_t selected_final_index = 0;  // 初始化采样结果

        // 处理特殊情况：如果总和过小或 k=0，则默认选择第一个 top-k 元素 (或 0)
        if (total_exp_sum <= 1e-9f || k == 0) {
            if (k > 0) {  // 如果 k>0 但总和接近 0，选择概率最高的那个
                selected_final_index = static_cast<uint32_t>(d_sorted_topk_indices[0]);
            } else {  // 如果 k=0 (理论上不应发生，因为前面有检查)，返回 0
                selected_final_index = 0;
            }
        } else {
            // 生成一个 [0, total_exp_sum) 范围内的随机数
            float r = hiprand_uniform(&localState) * total_exp_sum;
            float cumulative = 0.0f;  // 累积概率

            // 线性扫描共享内存中的 exp 值进行加权采样
            selected_final_index = static_cast<uint32_t>(d_sorted_topk_indices[0]);  // 默认值
            float* s_exp_vals = shared_storage.combined.exp_vals;                    // 指向共享内存中的 exp 数组
            for (int i = 0; i < k; ++i) {
                // 从共享内存读取预先计算好的 exp 值
                cumulative += s_exp_vals[i];
                // 如果累积和超过随机阈值 r，则选择当前索引
                if (cumulative >= r) {
                    selected_final_index = static_cast<uint32_t>(d_sorted_topk_indices[i]);
                    break;  // 找到后即退出循环
                }
            }
        }
        // 将最终选定的索引写入输出指针
        *d_sampled_index = selected_final_index;
        // 更新 cuRAND 状态
        states[0] = localState;
    }
}

// CUB TransformIterator 的辅助 Functor
// 功能: 在 CUB 操作中动态地将输入类型 Tin (如 bfloat16) 转换为 float。
template <typename Tin>
struct ConvertToFloatFunctor {
    __device__ __forceinline__ float operator()(const Tin& x) const {
        if constexpr (std::is_same_v<Tin, __hip_bfloat16>) {
            return __bfloat162float(x);  // bfloat16 转 float
        } else {
            return static_cast<float>(x);  // 其他类型直接转 float
        }
    }
};

// 主采样函数
// 功能: 对输入的 logits 执行 Top-K 采样。
// 输入:
//   - logits: 输入的 logits 张量 (T 类型, 形状 [seq_len, vocab_size], 必须在
//   CUDA 设备上)
//   - temperature: 温度系数，用于缩放 logits
//   - top_p: Top-P 采样的概率阈值 (当前代码中未使用)
//   - top_k: Top-K 采样的 K 值
//   - d_states: 指向设备端 cuRAND 状态的指针
//   - stream: CUDA 流
// 返回:
//   - 指向设备端存储最终采样结果 (单个 uint32_t 索引) 的指针。注意：结果仍在
//   GPU 上。
template <typename T>
uint32_t* sample(Tensor<T>&& logits, float temperature,
                 float top_p,  // top_p 未在此实现中使用
                 size_t top_k, hiprandState* d_states, hipStream_t stream) {
    // --- 输入验证 ---
    if (logits.device() != Device::CUDA) {
        throw std::runtime_error("输入张量必须在 CUDA 设备上");
    }
    // Top-K 采样至少需要 k=1
    if (top_k == 0) {
        throw std::runtime_error("top_k 必须至少为 1");
    }

    const auto& shape = logits.sizes();
    if (shape.size() != 2 || shape[0] == 0 || shape[1] == 0) {
        throw std::runtime_error("输入张量必须是二维且维度非零 [seq_len, vocab_size]");
    }

    const size_t seq_len = shape[0];
    const size_t vocab_size = shape[1];

    // 如果 K 大于词汇表大小，则将其限制为词汇表大小
    if (top_k > vocab_size) {
        top_k = vocab_size;
    }
    // 检查 K 是否超过 Kernel 2 中共享内存的限制 (MAX_TOPK)
    if (top_k > MAX_TOPK) {
        throw std::runtime_error("请求的 top_k (" + std::to_string(top_k) + ") 超过了 Kernel 2 的 MAX_TOPK 限制 (" +
                                 std::to_string(MAX_TOPK) + ")，无法在共享内存中分配。");
    }

    // 获取指向最后一个 token 的 logits 数据的设备指针
    const T* d_logits_ptr = logits.data_ptr() + (seq_len - 1) * vocab_size;

    // --- 内存管理 ---
    auto& pool = GlobalCudaMemoryPool::instance();  // 获取全局内存池实例
    // 使用 T 作为存储与排序键
    T* d_scaled_logits = static_cast<T*>(pool.allocate(vocab_size * sizeof(T)));         // 缩放后的 logits (T)
    float* d_max_val = static_cast<float*>(pool.allocate(sizeof(float)));                // 最大值
    int* d_indices = static_cast<int*>(pool.allocate(vocab_size * sizeof(int)));         // 原始索引
    T* d_sorted_logits = static_cast<T*>(pool.allocate(vocab_size * sizeof(T)));         // 排序后的 logits (T)
    int* d_sorted_indices = static_cast<int*>(pool.allocate(vocab_size * sizeof(int)));  // 排序后的索引

    // 使用tagged memory分配采样结果，确保每次都是相同的固定地址
    uint32_t* d_sampled_index = static_cast<uint32_t*>(pool.allocate_tagged("graph_input_token", sizeof(uint32_t)));

    // CUB 临时存储指针和大小 (初始为 nullptr 和 0)
    void* d_reduce_temp_storage = nullptr;
    size_t reduce_temp_storage_bytes = 0;
    void* d_sort_temp_storage = nullptr;
    size_t sort_temp_storage_bytes = 0;

    // 步骤 1 (融合 Kernel): 缩放 Logits 并初始化索引
    const int scale_init_block_size = 256;  // 定义块大小
    const int scale_init_grid_size =        // 计算网格大小
        (vocab_size + scale_init_block_size - 1) / scale_init_block_size;

    // 缩放到 T 并初始化索引
    scale_logits_and_init_indices_kernel<T><<<scale_init_grid_size, scale_init_block_size, 0, stream>>>(
        d_logits_ptr, d_scaled_logits, d_indices, vocab_size, temperature);
    CUDA_CHECK(hipGetLastError());  // 检查核函数启动错误

    // 步骤 2: 查找最大缩放 Logit (使用 CUB Device Reduce)
    // 转换迭代器：T -> float 求 Max
    hipcub::TransformInputIterator<float, ConvertToFloatFunctor<T>, const T*> itr(d_scaled_logits,
                                                                               ConvertToFloatFunctor<T>());
    CUDA_CHECK(
        hipcub::DeviceReduce::Max(d_reduce_temp_storage, reduce_temp_storage_bytes, itr, d_max_val, vocab_size, stream));
    // 分配所需的临时存储
    d_reduce_temp_storage = pool.allocate(reduce_temp_storage_bytes);
    // 第二次调用 CUB Reduce: 执行 Max 操作
    CUDA_CHECK(
        hipcub::DeviceReduce::Max(d_reduce_temp_storage, reduce_temp_storage_bytes, itr, d_max_val, vocab_size, stream));

    // 步骤 3: 按 Logit 值降序排序 (Logit, Index) 对 (使用 CUB Device Radix Sort)
    // 第一次调用 CUB Sort: 键为 T
    CUDA_CHECK(hipcub::DeviceRadixSort::SortPairsDescending(d_sort_temp_storage, sort_temp_storage_bytes, d_scaled_logits,
                                                         d_sorted_logits, d_indices, d_sorted_indices, vocab_size, 0,
                                                         sizeof(T) * 8, stream));
    // 分配所需的临时存储
    d_sort_temp_storage = pool.allocate(sort_temp_storage_bytes);
    // 第二次调用 CUB Sort: 执行排序操作
    CUDA_CHECK(hipcub::DeviceRadixSort::SortPairsDescending(d_sort_temp_storage, sort_temp_storage_bytes, d_scaled_logits,
                                                         d_sorted_logits, d_indices, d_sorted_indices, vocab_size, 0,
                                                         sizeof(T) * 8, stream));

    // 步骤 4: 从 Top-K 结果中进行最终加权采样 (单块 Kernel)
    // 为采样核函数选择块大小 (必须与核函数模板参数匹配)
    const int sample_block_size = 128;  // 示例块大小
    // 计算 Kernel 2 所需的共享内存大小
    // 需要 CUB Reduce 的临时存储大小 + 存储 exp 值的数组大小
    size_t reduce_storage_size_est = sizeof(hipcub::BlockReduce<float,
                                                             sample_block_size>::TempStorage);  // CUB Reduce 存储
    size_t exp_values_size = MAX_TOPK * sizeof(float);                     // 存储 exp 值的数组大小 (基于 MAX_TOPK)
    size_t sample_shared_mem = reduce_storage_size_est + exp_values_size;  // 总共享内存需求

    // 启动 Kernel 2 (单块，块大小为 sample_block_size)
    // 模板参数 <T, sample_block_size> 必须与核函数定义匹配
    sample_from_sorted_topk_kernel<T, sample_block_size><<<1, sample_block_size, sample_shared_mem, stream>>>(
        d_sorted_logits,   // 排序后的 top-k logits (T)
        d_sorted_indices,  // 排序后的 top-k indices
        top_k,             // 实际使用的 k 值 (已确保 <= MAX_TOPK)
        d_max_val,         // 最大 logit 值指针
        d_states,          // cuRAND 状态
        d_sampled_index    // 输出采样的索引
    );
    CUDA_CHECK(hipGetLastError());  // 检查核函数启动错误

    // 将所有临时分配的设备内存返还给内存池
    // 但其实意义不大
    pool.free(d_scaled_logits);
    pool.free(d_max_val);
    pool.free(d_indices);
    pool.free(d_sorted_logits);
    pool.free(d_sorted_indices);
    // pool.free(d_sampled_index); // 不释放返回值
    pool.free(d_reduce_temp_storage);
    pool.free(d_sort_temp_storage);

    // 返回指向设备端采样结果的指针
    return d_sampled_index;
}

// 高效采样函数的包装
template <typename T>
void fast_sample_to_fixed(Tensor<T>&& logits, uint32_t* output_ptr, float* /*prob_ptr*/, float temperature,
                          float /*top_p*/, size_t top_k, hiprandState* d_states, hipStream_t stream) {
    if (logits.device() != Device::CUDA) {
        throw std::runtime_error("输入张量必须在 CUDA 设备上");
    }

    const auto& shape = logits.sizes();
    if (shape.size() != 2 || shape[0] != 1) {
        throw std::runtime_error("输入张量必须是 [1, vocab_size] 形状");
    }

    const size_t vocab_size = shape[1];
    top_k = std::min(top_k, vocab_size);

    const T* d_logits_ptr = logits.data_ptr();

    auto& pool = GlobalCudaMemoryPool::instance();
    T* d_scaled_logits = static_cast<T*>(pool.allocate(vocab_size * sizeof(T)));
    float* d_max_val = static_cast<float*>(pool.allocate(sizeof(float)));
    int* d_indices = static_cast<int*>(pool.allocate(vocab_size * sizeof(int)));
    T* d_sorted_logits = static_cast<T*>(pool.allocate(vocab_size * sizeof(T)));
    int* d_sorted_indices = static_cast<int*>(pool.allocate(vocab_size * sizeof(int)));

    void* d_reduce_temp_storage = nullptr;
    size_t reduce_temp_storage_bytes = 0;
    void* d_sort_temp_storage = nullptr;
    size_t sort_temp_storage_bytes = 0;

    // 1) 缩放到 float 并初始化索引
    const int threads = 256;
    const int blocks = static_cast<int>((vocab_size + threads - 1) / threads);
    scale_logits_and_init_indices_kernel<T>
        <<<blocks, threads, 0, stream>>>(d_logits_ptr, d_scaled_logits, d_indices, vocab_size, temperature);
    CUDA_CHECK(hipGetLastError());

    // 2) Max
    hipcub::TransformInputIterator<float, ConvertToFloatFunctor<T>, const T*> itr2(d_scaled_logits,
                                                                                ConvertToFloatFunctor<T>());
    CUDA_CHECK(
        hipcub::DeviceReduce::Max(d_reduce_temp_storage, reduce_temp_storage_bytes, itr2, d_max_val, vocab_size, stream));
    d_reduce_temp_storage = pool.allocate(reduce_temp_storage_bytes);
    CUDA_CHECK(
        hipcub::DeviceReduce::Max(d_reduce_temp_storage, reduce_temp_storage_bytes, itr2, d_max_val, vocab_size, stream));

    // 3) SortPairsDescending (float keys)
    CUDA_CHECK(hipcub::DeviceRadixSort::SortPairsDescending(d_sort_temp_storage, sort_temp_storage_bytes, d_scaled_logits,
                                                         d_sorted_logits, d_indices, d_sorted_indices, vocab_size, 0,
                                                         sizeof(T) * 8, stream));
    d_sort_temp_storage = pool.allocate(sort_temp_storage_bytes);
    CUDA_CHECK(hipcub::DeviceRadixSort::SortPairsDescending(d_sort_temp_storage, sort_temp_storage_bytes, d_scaled_logits,
                                                         d_sorted_logits, d_indices, d_sorted_indices, vocab_size, 0,
                                                         sizeof(T) * 8, stream));

    // 4) 采样（单块）
    const int sample_block_size = 128;
    size_t sample_shared_mem =
        sizeof(hipcub::BlockReduce<float, sample_block_size>::TempStorage) + MAX_TOPK * sizeof(float);
    sample_from_sorted_topk_kernel<T, sample_block_size><<<1, sample_block_size, sample_shared_mem, stream>>>(
        d_sorted_logits, d_sorted_indices, top_k, d_max_val, d_states, output_ptr);
    CUDA_CHECK(hipGetLastError());

    pool.free(d_scaled_logits);
    pool.free(d_max_val);
    pool.free(d_indices);
    pool.free(d_sorted_logits);
    pool.free(d_sorted_indices);
    pool.free(d_reduce_temp_storage);
    pool.free(d_sort_temp_storage);
}
// 并行批量采样核函数 - 避免for循环瓶颈
template <typename T>
__global__ void sample_batch_parallel_kernel(
    const T* __restrict__ logits_data,  // 输入: 批量logits数据 [seq_len, vocab_size]
    uint32_t* output_ptr,               // 输出: 采样结果数组 [seq_len]
    size_t seq_len,                     // 序列长度
    size_t vocab_size,                  // 词汇表大小
    float temperature,                  // 温度参数
    size_t top_k,                       // Top-K采样参数
    hiprandState* d_states               // 随机数状态
) {
    // 每个线程块处理一个序列位置
    int seq_idx = blockIdx.x;
    if (seq_idx >= seq_len)
        return;

    int tid = threadIdx.x;
    int block_size = blockDim.x;

    // 共享内存用于存储处理结果
    extern __shared__ float shared_mem[];
    float* s_scaled_logits = shared_mem;  // 缩放后的logits
    // float* s_exp_vals = s_scaled_logits + vocab_size;  // exp值
    // int* s_indices = (int*)(s_exp_vals + top_k);               // 索引数组

    // 获取当前序列位置的logits指针
    const T* seq_logits = logits_data + seq_idx * vocab_size;

    // 第一步：缩放logits并找到最大值
    float max_val = -FLT_MAX;

    // 并行处理所有词汇，找到最大值
    for (int i = tid; i < vocab_size; i += block_size) {
        float logit_f;
        if constexpr (std::is_same_v<T, __hip_bfloat16>) {
            logit_f = __bfloat162float(seq_logits[i]);
        } else {
            logit_f = static_cast<float>(seq_logits[i]);
        }
        s_scaled_logits[i] = logit_f / temperature;
        max_val = fmaxf(max_val, s_scaled_logits[i]);
    }

    // 块内归约找到全局最大值
    __shared__ float s_max_val;
    __syncthreads();

    // 简单的归约操作
    for (int stride = block_size / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            max_val = fmaxf(max_val, __shfl_down_sync(0xFFFFFFFF, max_val, stride));
        }
    }
    if (tid == 0) {
        s_max_val = max_val;
    }
    __syncthreads();

    // 第二步：计算exp值并构建top-k
    float exp_sum = 0.0f;

    // 并行计算exp值
    for (int i = tid; i < vocab_size; i += block_size) {
        float exp_val = expf(s_scaled_logits[i] - s_max_val);
        s_scaled_logits[i] = exp_val;  // 复用shared memory存储exp值
        exp_sum += exp_val;
    }

    // 归约求和
    for (int stride = block_size / 2; stride > 0; stride /= 2) {
        exp_sum += __shfl_down_sync(0xFFFFFFFF, exp_sum, stride);
    }

    __shared__ float s_exp_sum;
    if (tid == 0) {
        s_exp_sum = exp_sum;
    }
    __syncthreads();

    // 第三步：简化的top-k采样（使用简单的线性搜索代替完整排序）
    if (tid == 0) {
        hiprandState local_state = d_states[seq_idx % 1];  // 简化：复用状态

        float r = hiprand_uniform(&local_state) * s_exp_sum;
        float cumulative = 0.0f;
        uint32_t selected_idx = 0;

        // 线性扫描找到采样位置
        for (int i = 0; i < vocab_size; ++i) {
            cumulative += s_scaled_logits[i];
            if (cumulative >= r) {
                selected_idx = i;
                break;
            }
        }

        output_ptr[seq_idx] = selected_idx;
        d_states[seq_idx % 1] = local_state;
    }
}

// 批量采样函数的变体，将结果写入指定的GPU内存位置数组 - 优化版本
template <typename T>
void sample_batch_to_fixed(Tensor<T>&& logits, uint32_t* output_ptr, float temperature, float top_p, size_t top_k,
                           hiprandState* d_states, hipStream_t stream) {
    if (logits.device() != Device::CUDA) {
        throw std::runtime_error("输入张量必须在 CUDA 设备上");
    }
    // Top-K 采样至少需要 k=1
    if (top_k == 0) {
        throw std::runtime_error("top_k 必须至少为 1");
    }

    const auto& shape = logits.sizes();
    if (shape.size() != 2 || shape[0] == 0 || shape[1] == 0) {
        throw std::runtime_error("输入张量必须是二维且维度非零 [seq_len, vocab_size]");
    }

    const size_t seq_len = shape[0];
    const size_t vocab_size = shape[1];

    // 回退到原始实现（如果共享内存不足）
    for (size_t i = 0; i < seq_len; i++) {
        // 创建当前位置logits的视图
        std::vector<size_t> start = {i, 0};
        std::vector<size_t> end = {i + 1, vocab_size};
        Tensor<T> logit_view = logits.slice(start, end);

        // 调用单个token的sample_to_fixed函数，将结果写入output_ptr[i]
        sample_to_fixed(std::move(logit_view), output_ptr + i, temperature, top_p, top_k, d_states, stream);
    }
}

// 采样函数的变体，将结果写入指定的GPU内存位置
template <typename T>
void sample_to_fixed(Tensor<T>&& logits, uint32_t* output_ptr, float temperature, float top_p, size_t top_k,
                     hiprandState* d_states, hipStream_t stream) {
    // 使用优化后的fast_sample_to_fixed函数
    fast_sample_to_fixed(std::move(logits), output_ptr, nullptr, temperature, top_p, top_k, d_states, stream);
}

template uint32_t* sample<float>(Tensor<float>&&, float, float, size_t, hiprandState*, hipStream_t);
template uint32_t* sample<__hip_bfloat16>(Tensor<__hip_bfloat16>&&, float, float, size_t, hiprandState*, hipStream_t);

template void sample_to_fixed<float>(Tensor<float>&&, uint32_t*, float, float, size_t, hiprandState*, hipStream_t);
template void sample_to_fixed<__hip_bfloat16>(Tensor<__hip_bfloat16>&&, uint32_t*, float, float, size_t, hiprandState*,
                                             hipStream_t);

template void sample_batch_to_fixed<float>(Tensor<float>&&, uint32_t*, float, float, size_t, hiprandState*,
                                           hipStream_t);
template void sample_batch_to_fixed<__hip_bfloat16>(Tensor<__hip_bfloat16>&&, uint32_t*, float, float, size_t,
                                                   hiprandState*, hipStream_t);

template void fast_sample_to_fixed<float>(Tensor<float>&&, uint32_t*, float*, float, float, size_t, hiprandState*,
                                          hipStream_t);
template void fast_sample_to_fixed<__hip_bfloat16>(Tensor<__hip_bfloat16>&&, uint32_t*, float*, float, float, size_t,
                                                  hiprandState*, hipStream_t);

}  // namespace cuda_OP

#endif  // CUDA_OP_CUH