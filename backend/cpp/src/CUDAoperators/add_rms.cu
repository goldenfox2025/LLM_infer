#include "hip/hip_runtime.h"
#include <hip/hip_bf16.h> // Required for hip_bfloat16, nv_bfloat162, __hadd2 (though hadd2 is unused correctly now)
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h> // Required for printf

#include <algorithm> // For std::min
#include <cstdio>
#include <iostream>
#include <stdexcept>   // For std::runtime_error
#include <type_traits> // For std::is_same_v
#include <vector>

// Assume cudaOP.cuh contains Tensor definition, checkCudaError, etc.
#include "cudaOP.cuh"

namespace cuda_OP
{
    // warp_reduce_sum 函数保持不变
    __device__ inline float warp_reduce_sum(float val)
    {
        for (int offset = 32 / 2; offset > 0; offset /= 2)
        {
            val += __shfl_down_sync(__activemask(), val, offset);
        }
        return val;
    }

    template <typename T>
    __global__ void add_rms_kernel(
        T *output,
        T *input,
        const T *add_,
        const T *weight, float eps,
        size_t row_size)
    {
        // 获取当前 block 和 thread 的 ID
        int row = blockIdx.x;
        int tid = threadIdx.x;
        int nthreads = blockDim.x;
        int lane = tid % warpSize;
        int warp_id = tid / warpSize;

        T *in_row = input + row * row_size;
        T *out_row = output + row * row_size;
        const T *add_row = add_ + row * row_size;
        float val[5];
        int flag = 0;
        float local_sum = 0.0f;
        for (size_t i = tid; i < row_size; i += nthreads)
        {


            val[flag++] = static_cast<float>(in_row[i] + add_row[i]);
          

            // 累加平方和
            local_sum += val[flag-1] * val[flag-1];
        }

   
        float warp_sum = warp_reduce_sum(local_sum);


        __shared__ float s_warp_sums[32]; 

     
        if (lane == 0)
        {

            s_warp_sums[warp_id] = warp_sum;
        }
        __syncthreads();

        float block_sum = 0.0f;
        if (warp_id == 0)
        {
            int num_warps_in_block = (nthreads + warpSize - 1) / warpSize;

            float warp_partial_sum = (tid < num_warps_in_block) ? s_warp_sums[tid] : 0.0f;

            // 在 Warp 0 内进行最终的归约
            block_sum = warp_reduce_sum(warp_partial_sum);

            // ---
        }

        __shared__ float s_inv_rms;
        if (tid == 0)
        {
            float mean_sq = block_sum / row_size;
            float rsqrt_arg = mean_sq + eps;
            s_inv_rms = rsqrtf(rsqrt_arg);
        }
        __syncthreads(); // 确保所有线程都能读到 s_inv_rms

        // 5. 应用 RMSNorm: output = (input + add) * inv_rms * weight
        float inv_rms = s_inv_rms; // 所有线程获取计算好的 inv_rms
        flag = 0;
        for (size_t i = tid; i < row_size; i += nthreads)
        {
            if (i < row_size) // 边界检查
            {
                // 重新计算 val = input + add (或者从 shared memory 读取，如果做了优化)
                // float val_f = static_cast<float>(in_row[i]);

                // float add_f = static_cast<float>(add_row[i]);

                // float val = static_cast<float>(in_row[i] + add_row[i]);

                // 读取 weight
                float w = static_cast<float>(weight[i]);

                // 计算最终结果
                float normalized_val = val[flag++] * inv_rms;
                float scaled_val = normalized_val * w;

                out_row[i] = static_cast<T>(scaled_val);
                in_row[i] = static_cast<T>(val[flag-1]); 
            }
        }
    }


    template <typename T>
    void add_rms(Tensor<T> *output,  Tensor<T> *input, const Tensor<T> *add_,
                 const Tensor<T> *weight, float eps, hipStream_t stream)
    {
        size_t seq_len = input->sizes()[0];
        size_t d = input->sizes()[1]; // row_size

        int threads_per_block = 1024; 
        // 简单的启发式：如果 d 较小，减少线程数以避免浪费
        if (d < 1024)
        {

            if (d <= 32)
                threads_per_block = 32;
            else if (d <= 64)
                threads_per_block = 64;
            else if (d <= 128)
                threads_per_block = 128;
            else if (d <= 256)
                threads_per_block = 256;
            else if (d <= 512)
                threads_per_block = 512;
            else
                threads_per_block = 1024;
        }

        dim3 block_dim(threads_per_block);
        dim3 grid_dim(seq_len);

        // --- 调用 Debug Kernel ---
        add_rms_kernel<T><<<grid_dim, block_dim, 0, stream>>>(
            output->data_ptr(), input->data_ptr(), add_->data_ptr(), weight->data_ptr(), eps, d);
        // ---

        checkCudaError(hipGetLastError());

    }

    // 模板实例化保持不变
    template void add_rms<float>(Tensor<float> *,  Tensor<float> *, const Tensor<float> *,
                                 const Tensor<float> *, float, hipStream_t);
    template void add_rms<nvbf16>(Tensor<nvbf16> *,  Tensor<nvbf16> *, const Tensor<nvbf16> *,
                                  const Tensor<nvbf16> *, float, hipStream_t);

} // namespace cuda_OP