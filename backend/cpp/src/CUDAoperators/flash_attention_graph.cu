#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <math.h>

#include <algorithm>
#include <cstdio>
#include <cstring>
#include <iostream>
#include <stdexcept>
#include <vector>

#include "cudaOP.cuh"

#define DQKV_VALUE 128
#define B_C_VALUE 8
#define MAX_BRANCHES 5

constexpr int WARP_SIZE = 32;

namespace cuda_OP {

// CUDA图优化版本的flash attention kernel
// 直接从连续的KV缓存中读取数据，仿照flash_attention_variable的模式
template <typename T>
__global__ void flash_attention_kernel_graph_fixed(
    T *q,
    const T *total_k,    // 连续的K缓存 [total_seq_len, n_kv_h, dqkv]
    const T *total_v,    // 连续的V缓存 [total_seq_len, n_kv_h, dqkv]
    T **output_ptrs,     // 固定的输出指针数组
    int *segment_info,   // 分段信息：[total_seq_len, branch_count, branch_lengths...]
    int n_q_h, int n_kv_h, int dqkv, int B_c, int B_r, int n_groups, int T_r,
    T softmax_scale) {

  // 从设备内存读取分段信息
  int total_seq_len = segment_info[0];
  // segment_info[1] (active_branches) 已经无用，始终使用固定3分支

  // 固定使用3分支模式，blockIdx.y就是分支索引(0,1,2)
  const int FIXED_BRANCHES = 3;

  // 检查分支ID是否有效
  if (blockIdx.y >= FIXED_BRANCHES) return;

  // 关键修复：使用与普通推理完全相同的分段逻辑
  // 普通推理使用：tokens_per_branch = (total_seq_len + branches_needed - 1) / branches_needed
  // 这是向上取整的除法，确保所有token都被覆盖
  int branches_needed = FIXED_BRANCHES;
  int tokens_per_branch = (total_seq_len + branches_needed - 1) / branches_needed;

  int start_idx, end_idx;
  if (blockIdx.y == 0) {
    start_idx = 0;
    end_idx = min(tokens_per_branch, total_seq_len);
  } else if (blockIdx.y == 1) {
    start_idx = tokens_per_branch;
    end_idx = min(2 * tokens_per_branch, total_seq_len);
  } else { // blockIdx.y == 2
    start_idx = 2 * tokens_per_branch;
    end_idx = total_seq_len;
  }

  int cache_length = end_idx - start_idx;

  // 如果分支长度为0，直接退出
  if (cache_length <= 0) return;

  int T_c = (cache_length + B_c - 1) / B_c;
  T *att_output = output_ptrs[blockIdx.y];

  // 验证参数
  if (dqkv != DQKV_VALUE || B_c != B_C_VALUE) return;

  __shared__ float s_qi[DQKV_VALUE];
  __shared__ float s_vj[B_C_VALUE * DQKV_VALUE];
  __shared__ float s_score_buf[B_C_VALUE];
  __shared__ float s_lm[2];
  __shared__ float s_s_score[B_C_VALUE];
  __shared__ float s_o[DQKV_VALUE];

  const int d_tid = threadIdx.x;
  const int token_tid = threadIdx.y;
  const int head_id = blockIdx.x;
  const int q_offset = head_id * dqkv;
  const int kv_head = head_id / n_groups;

  constexpr int vec_unit = 16 / sizeof(T);
  Vec<T, vec_unit> vq, vk, vv;

  const int vecCount = dqkv / vec_unit;
  for (int i = d_tid; i < vecCount; i += blockDim.x) {
    vq.f4 = *reinterpret_cast<const float4 *>(&q[q_offset + i * vec_unit]);
    if (token_tid < vec_unit)
      s_qi[i * vec_unit + token_tid] = static_cast<float>(vq.t[token_tid]);
  }
  __syncthreads();

  float &global_m = s_lm[0];
  float &global_l = s_lm[1];

  // 遍历 KV 分块
  for (int j = 0; j < T_c; ++j) {
    int token_index = j * B_c + token_tid;
    bool valid = (token_index < cache_length);
    float local_score = 0.0f;

    for (int i = d_tid; i < vecCount; i += blockDim.x) {
      // 关键修复：正确计算在整个KV cache中的绝对索引
      // token_index是分支内相对索引，需要加上start_idx得到绝对位置
      int absolute_token_idx = start_idx + token_index;

      // 关键修复：确保不会越界访问
      if (valid && absolute_token_idx < total_seq_len) {
        int index = (absolute_token_idx * n_kv_h + kv_head) * dqkv + i * vec_unit;
        vk.f4 = *reinterpret_cast<const float4 *>(&total_k[index]);
        vv.f4 = *reinterpret_cast<const float4 *>(&total_v[index]);
#pragma unroll
        for (int l = 0; l < vec_unit; l++) {
          float k_val = static_cast<float>(vk.t[l]);
          float v_val = static_cast<float>(vv.t[l]);
          local_score += s_qi[i * vec_unit + l] * k_val;
          s_vj[token_tid * DQKV_VALUE + i * vec_unit + l] = v_val;
        }
      } else {
#pragma unroll
        for (int l = 0; l < vec_unit; l++) {
          s_vj[token_tid * DQKV_VALUE + i * vec_unit + l] = 0.0f;
        }
      }
    }

    __syncthreads();

    // Warp 内归约 QK Score
    // 关键修复：需要同时检查分支内有效性和绝对索引有效性
    int absolute_token_idx = start_idx + token_index;
    bool absolutely_valid = valid && (absolute_token_idx < total_seq_len);

    if (absolutely_valid) {
      unsigned int mask = 0xFFFFFFFF;
      for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        local_score += __shfl_down_sync(mask, local_score, offset);
      }
      if (d_tid == 0) {
        s_score_buf[token_tid] = local_score * static_cast<float>(softmax_scale);
      }
    } else {
      if (d_tid == 0) {
        s_score_buf[token_tid] = -FLT_MAX;
      }
    }
    __syncthreads();

    // Local Softmax
    __shared__ float cur_m_s;
    float warp_val = (d_tid < B_c && threadIdx.y == 0) ? s_score_buf[d_tid] : -FLT_MAX;
    unsigned int mask_max = 0xFFFFFFFF;
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
      warp_val = fmaxf(warp_val, __shfl_down_sync(mask_max, warp_val, offset));
    }
    if (d_tid == 0 && threadIdx.y == 0) {
      cur_m_s = warp_val;
    }
    __syncthreads();
    float cur_m = cur_m_s;

    __shared__ float cur_l_s;
    float warp_val_l = 0.0f;
    if (d_tid < B_c && threadIdx.y == 0) {
      float score_val = s_score_buf[d_tid];
      float exp_val = expf(score_val - cur_m);
      s_s_score[d_tid] = exp_val;
      warp_val_l = exp_val;
    } else {
      warp_val_l = 0.0f;
    }

    __syncthreads();

    // 求和归约
    unsigned int mask_sum = 0xFFFFFFFF;
    for (int offset = WARP_SIZE / 2; offset > 0; offset >>= 1) {
      warp_val_l += __shfl_down_sync(mask_sum, warp_val_l, offset);
    }
    if (d_tid == 0 && threadIdx.y == 0) {
      cur_l_s = warp_val_l;
    }
    __syncthreads();
    float cur_l = cur_l_s;

    // 计算部分输出
    if (j == 0) {
      // 第一个块: 计算并直接写入 s_o
      if (token_tid == 0) {
        for (int k_dim = d_tid; k_dim < DQKV_VALUE; k_dim += blockDim.x) {
          float current_dim_partial_out = 0.0f;
          for (int i_tok = 0; i_tok < B_c; ++i_tok) {
            float exp_score = s_s_score[i_tok];
            float v_val = s_vj[i_tok * DQKV_VALUE + k_dim];
            current_dim_partial_out = fmaf(exp_score, v_val, current_dim_partial_out);
          }
          s_o[k_dim] = current_dim_partial_out;
        }
      }
      if (token_tid == 0 && d_tid == 0) {
        global_m = cur_m;
        global_l = cur_l;
      }
    } else {
      // 后续块: Online update
      float old_global_m = global_m;
      float old_global_l = global_l;
      float new_global_m = fmaxf(old_global_m, cur_m);
      float exp_old = __expf(old_global_m - new_global_m);
      float exp_cur = __expf(cur_m - new_global_m);

      if (token_tid == 0) {
        for (int k_dim = d_tid; k_dim < DQKV_VALUE; k_dim += blockDim.x) {
          float current_dim_partial_out = 0.0f;
          for (int i_tok = 0; i_tok < B_c; ++i_tok) {
            float exp_score = s_s_score[i_tok];
            float v_val = s_vj[i_tok * DQKV_VALUE + k_dim];
            current_dim_partial_out = fmaf(exp_score, v_val, current_dim_partial_out);
          }
          float old_out_val = s_o[k_dim];
          float new_out_val = old_out_val * exp_old + current_dim_partial_out * exp_cur;
          s_o[k_dim] = new_out_val;
        }
      }

      if (token_tid == 0 && d_tid == 0) {
        float new_global_l = old_global_l * exp_old + cur_l * exp_cur;
        global_m = new_global_m;
        global_l = new_global_l;
      }
    }
    __syncthreads();
  }

  // 写回 att_output
  if (threadIdx.y == 0) {
    int out_offset = head_id * (dqkv + 2);
    for (int i = d_tid; i < DQKV_VALUE; i += blockDim.x) {
      att_output[out_offset + i] = static_cast<T>(s_o[i]);
    }
    if (d_tid == 0) {
      att_output[out_offset + dqkv] = static_cast<T>(global_m);
      att_output[out_offset + dqkv + 1] = static_cast<T>(global_l);
    }
  }
}

// CUDA图优化版本：使用固定内存地址和分段信息的flash attention
// 仿照dynamic_flash_attention_wrapper的模式，直接接受连续的KV缓存
template <typename T>
void flash_attention_graph_fixed(Tensor<T> &Q,
                                 const Tensor<T> &total_K,
                                 const Tensor<T> &total_V,
                                 T **d_output_ptrs,
                                 int *d_segment_info,
                                 int n_kv_heads,
                                 hipStream_t stream) {

  int dqkv = Q.sizes()[2];
  if (dqkv != DQKV_VALUE) {
    throw std::runtime_error("dqkv 不匹配预定义的值");
  }

  float softmax_scale = 1.0f / sqrtf(static_cast<float>(dqkv));
  int n_q_h = Q.sizes()[1];
  int n_groups = n_q_h / n_kv_heads;
  int B_r = 1;
  int T_r = 1;
  int B_c = B_C_VALUE;

  // 设置kernel参数 - 强制使用3分支，类似flash_attention.cu的稳定模式
  const int FIXED_BRANCHES = 3;
  dim3 grid(n_q_h, FIXED_BRANCHES);
  dim3 block(32, B_c);

  // 启动kernel
  flash_attention_kernel_graph_fixed<T><<<grid, block, 0, stream>>>(
      Q.data_ptr(),
      total_K.data_ptr(),
      total_V.data_ptr(),
      d_output_ptrs,
      d_segment_info,
      n_q_h, n_kv_heads, dqkv, B_c, B_r, n_groups, T_r,
      static_cast<T>(softmax_scale));

  // 检查错误
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    throw std::runtime_error("CUDA error in flash_attention_graph_fixed: " +
                            std::string(hipGetErrorString(err)));
  }
}

// 显式模板实例化
template void flash_attention_graph_fixed<float>(
    Tensor<float> &Q,
    const Tensor<float> &total_K,
    const Tensor<float> &total_V,
    float **d_output_ptrs,
    int *d_segment_info,
    int n_kv_heads,
    hipStream_t stream);

template void flash_attention_graph_fixed<__hip_bfloat16>(
    Tensor<__hip_bfloat16> &Q,
    const Tensor<__hip_bfloat16> &total_K,
    const Tensor<__hip_bfloat16> &total_V,
    __hip_bfloat16 **d_output_ptrs,
    int *d_segment_info,
    int n_kv_heads,
    hipStream_t stream);

}  // namespace cuda_OP
