#include "hip/hip_runtime.h"
// #define MULTIST

#ifdef MULTIST
#include <hipblas.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <math.h>

#include <algorithm>
#include <cstdio>
#include <cstring>
#include <iostream>
#include <mutex>
#include <stdexcept>
#include <vector>

#include "cudaOP.cuh"
#include "ptx_common.h"
#define DQKV_VALUE 128
#define B_C_VALUE 16

namespace cuda_OP
{

    template <typename T>
    __device__ void compute_attention_block(int j, int compute_stage_idx, int cache_length, int start_idx,
                                            int total_seq_len, int dqkv, int B_c, T softmax_scale, const float *s_qi,
                                            const T *s_kj, const T *s_vj, float *s_score_buf, float *s_lm, float *s_s_score,
                                            float *s_o)
    {
        const int d_tid = threadIdx.x;
        const int token_tid = threadIdx.y;

        float local_score = 0.0f;
        for (int k = d_tid; k < dqkv; k += blockDim.x)
        {
            float q_val = s_qi[k];
            float k_val = static_cast<float>(s_kj[compute_stage_idx * B_c * dqkv + token_tid * dqkv + k]);
            local_score += q_val * k_val;
        }

        unsigned int mask = 0xFFFFFFFF;
        for (int offset = warpSize / 2; offset > 0; offset /= 2)
        {
            local_score += __shfl_down_sync(mask, local_score, offset);
        }

        int token_index = j * B_c + token_tid;
        bool valid = (token_index < cache_length);
        int absolute_token_idx = start_idx + token_index;
        bool absolutely_valid = valid && (absolute_token_idx < total_seq_len);

        if (d_tid == 0)
        {
            if (absolutely_valid)
            {
                s_score_buf[token_tid] = local_score * static_cast<float>(softmax_scale);
            }
            else
            {
                s_score_buf[token_tid] = -FLT_MAX;
            }
        }
        __syncthreads();

        __shared__ float cur_m_s;
        float warp_val = (d_tid < B_c && threadIdx.y == 0) ? s_score_buf[d_tid] : -FLT_MAX;
        unsigned int mask_max = 0xFFFFFFFF;
        for (int offset = warpSize / 2; offset > 0; offset /= 2)
        {
            warp_val = fmaxf(warp_val, __shfl_down_sync(mask_max, warp_val, offset));
        }
        if (d_tid == 0 && threadIdx.y == 0)
        {
            cur_m_s = warp_val;
        }
        __syncthreads();
        float cur_m = cur_m_s;

        __shared__ float cur_l_s;
        float warp_val_l = 0.0f;
        if (d_tid < B_c && threadIdx.y == 0)
        {
            float score_val = s_score_buf[d_tid];
            float exp_val = expf(score_val - cur_m);
            s_s_score[d_tid] = exp_val;
            warp_val_l = exp_val;
        }
        else
        {
            warp_val_l = 0.0f;
        }
        __syncthreads();

        unsigned int mask_sum = 0xFFFFFFFF;
        for (int offset = warpSize / 2; offset > 0; offset >>= 1)
        {
            warp_val_l += __shfl_down_sync(mask_sum, warp_val_l, offset);
        }
        if (d_tid == 0 && threadIdx.y == 0)
        {
            cur_l_s = warp_val_l;
        }
        __syncthreads();
        float cur_l = cur_l_s;

        float &global_m = s_lm[0];
        float &global_l = s_lm[1];

        if (j == 0)
        {
            if (token_tid == 0)
            {
                for (int k_dim = d_tid; k_dim < dqkv; k_dim += blockDim.x)
                {
                    float current_dim_partial_out = 0.0f;
                    for (int i_tok = 0; i_tok < B_c; ++i_tok)
                    {
                        float exp_score = s_s_score[i_tok];
                        float v_val = static_cast<float>(s_vj[compute_stage_idx * B_c * dqkv + i_tok * dqkv + k_dim]);
                        current_dim_partial_out = fmaf(exp_score, v_val, current_dim_partial_out);
                    }
                    s_o[k_dim] = current_dim_partial_out;
                }
            }
            if (token_tid == 0 && d_tid == 0)
            {
                global_m = cur_m;
                global_l = cur_l;
            }
        }
        else
        {
            float old_global_m = global_m;
            float old_global_l = global_l;
            float new_global_m = fmaxf(old_global_m, cur_m);
            float exp_old = __expf(old_global_m - new_global_m);
            float exp_cur = __expf(cur_m - new_global_m);

            if (token_tid == 0)
            {
                for (int k_dim = d_tid; k_dim < dqkv; k_dim += blockDim.x)
                {
                    float current_dim_partial_out = 0.0f;
                    for (int i_tok = 0; i_tok < B_c; ++i_tok)
                    {
                        float exp_score = s_s_score[i_tok];
                        float v_val = static_cast<float>(s_vj[compute_stage_idx * B_c * dqkv + i_tok * dqkv + k_dim]);
                        current_dim_partial_out = fmaf(exp_score, v_val, current_dim_partial_out);
                    }
                    float old_out_val = s_o[k_dim];
                    float new_out_val = old_out_val * exp_old + current_dim_partial_out * exp_cur;
                    s_o[k_dim] = new_out_val;
                }
            }

            if (token_tid == 0 && d_tid == 0)
            {
                float new_global_l = old_global_l * exp_old + cur_l * exp_cur;
                global_m = new_global_m;
                global_l = new_global_l;
            }
        }
        __syncthreads();
    }

    template <typename T, int STAGE = 2>
    __global__ void flash_attention_kernel_graph_fixed(T *q, const T *total_k, const T *total_v, T **output_ptrs,
                                                       int *segment_info, int n_q_h, int n_kv_h, int dqkv, int B_c, int B_r,
                                                       int n_groups, int T_r, T softmax_scale, int *pingpong_index)
    {
        int total_seq_len = segment_info[*pingpong_index];
        const int FIXED_BRANCHES = 3;

        if (blockIdx.y >= FIXED_BRANCHES)
            return;

        int branches_needed = FIXED_BRANCHES;
        int tokens_per_branch = (total_seq_len + branches_needed - 1) / branches_needed;

        int start_idx, end_idx;
        if (blockIdx.y == 0)
        {
            start_idx = 0;
            end_idx = min(tokens_per_branch, total_seq_len);
        }
        else if (blockIdx.y == 1)
        {
            start_idx = tokens_per_branch;
            end_idx = min(2 * tokens_per_branch, total_seq_len);
        }
        else
        { // blockIdx.y == 2
            start_idx = 2 * tokens_per_branch;
            end_idx = total_seq_len;
        }

        int cache_length = end_idx - start_idx;
        if (cache_length <= 0)
            return;

        int T_c = (cache_length + B_c - 1) / B_c;
        T *att_output = output_ptrs[blockIdx.y];

        if (dqkv != DQKV_VALUE || B_c != B_C_VALUE)
            return;

        extern __shared__ char s_mem[];
        float *s_qi = (float *)s_mem;
        T *s_kj = (T *)(s_qi + DQKV_VALUE);
        T *s_vj = (T *)(s_kj + STAGE * B_C_VALUE * DQKV_VALUE);
        float *s_score_buf = (float *)((char *)s_vj + STAGE * B_C_VALUE * DQKV_VALUE * sizeof(T));
        float *s_lm = (float *)(s_score_buf + B_C_VALUE);
        float *s_s_score = (float *)(s_lm + 2);
        float *s_o = (float *)(s_s_score + B_C_VALUE);

        const int d_tid = threadIdx.x;
        const int tid = threadIdx.y * blockDim.x + d_tid;
        const int num_threads = blockDim.x * blockDim.y;
        const int head_id = blockIdx.x;
        const int q_offset = head_id * dqkv;
        const int kv_head = head_id / n_groups;
        constexpr int vec_unit = 16 / sizeof(T);

        if (threadIdx.y == 0)
        {
            for (int i = d_tid; i < dqkv; i += blockDim.x)
            {
                s_qi[i] = static_cast<float>(q[q_offset + i]);
            }
        }
        __syncthreads();

        for (int i = 0; i < STAGE - 1; ++i)
        {
            int token_index_base = i * B_c;
            if (token_index_base >= cache_length)
                continue;
            for (int load_idx = tid * vec_unit; load_idx < B_c * dqkv; load_idx += num_threads * vec_unit)
            {
                int smem_row = load_idx / dqkv;
                int dim_idx = load_idx % dqkv;
                int absolute_token_idx = start_idx + token_index_base + smem_row;
                bool valid = (token_index_base + smem_row < cache_length) && (absolute_token_idx < total_seq_len);

                const T *k_ptr = total_k + (absolute_token_idx * n_kv_h + kv_head) * dqkv + dim_idx;
                const T *v_ptr = total_v + (absolute_token_idx * n_kv_h + kv_head) * dqkv + dim_idx;

                uint32_t k_smem_cp = __cvta_generic_to_shared(&s_kj[i * B_c * dqkv + smem_row * dqkv + dim_idx]);
                uint32_t v_smem_cp = __cvta_generic_to_shared(&s_vj[i * B_c * dqkv + smem_row * dqkv + dim_idx]);

                if (valid)
                {
                    CP_ASYNC_CG(k_smem_cp, k_ptr, 16);
                    CP_ASYNC_CG(v_smem_cp, v_ptr, 16);
                }
                else
                {
                    // Directly write zeros to shared memory instead of loading from global memory.
                    // This avoids the non-coalesced access to zero_vec.
                    float4 zero_val = {0.0f, 0.0f, 0.0f, 0.0f};
                    *reinterpret_cast<float4 *>(&s_kj[i * B_c * dqkv + smem_row * dqkv + dim_idx]) = zero_val;
                    *reinterpret_cast<float4 *>(&s_vj[i * B_c * dqkv + smem_row * dqkv + dim_idx]) = zero_val;
                }
            }
            CP_ASYNC_COMMIT_GROUP();
        }

        int main_loop_iters = (T_c < STAGE) ? 0 : (T_c - (STAGE - 1));
        for (int j = 0; j < main_loop_iters; ++j)
        {
            int j_load = j + STAGE - 1;
            int compute_stage_idx = j % STAGE;
            int load_stage_idx = j_load % STAGE;

            CP_ASYNC_WAIT_GROUP(STAGE - 2);
            __syncthreads();

            int token_index_base_load = j_load * B_c;
            if (token_index_base_load < cache_length)
            {
                for (int load_idx = tid * vec_unit; load_idx < B_c * dqkv; load_idx += num_threads * vec_unit)
                {
                    int smem_row = load_idx / dqkv;
                    int dim_idx = load_idx % dqkv;
                    int absolute_token_idx = start_idx + token_index_base_load + smem_row;
                    bool valid = (token_index_base_load + smem_row < cache_length) && (absolute_token_idx < total_seq_len);

                    const T *k_ptr = total_k + (absolute_token_idx * n_kv_h + kv_head) * dqkv + dim_idx;
                    const T *v_ptr = total_v + (absolute_token_idx * n_kv_h + kv_head) * dqkv + dim_idx;

                    uint32_t k_smem_cp =
                        __cvta_generic_to_shared(&s_kj[load_stage_idx * B_c * dqkv + smem_row * dqkv + dim_idx]);
                    uint32_t v_smem_cp =
                        __cvta_generic_to_shared(&s_vj[load_stage_idx * B_c * dqkv + smem_row * dqkv + dim_idx]);

                    if (valid)
                    {
                        CP_ASYNC_CG(k_smem_cp, k_ptr, 16);
                        CP_ASYNC_CG(v_smem_cp, v_ptr, 16);
                    }
                    else
                    {
                        // Directly write zeros to shared memory instead of loading from global memory.
                        float4 zero_val = {0.0f, 0.0f, 0.0f, 0.0f};
                        *reinterpret_cast<float4 *>(&s_kj[load_stage_idx * B_c * dqkv + smem_row * dqkv + dim_idx]) = zero_val;
                        *reinterpret_cast<float4 *>(&s_vj[load_stage_idx * B_c * dqkv + smem_row * dqkv + dim_idx]) = zero_val;
                    }
                }
                CP_ASYNC_COMMIT_GROUP();
            }

            compute_attention_block(j, compute_stage_idx, cache_length, start_idx, total_seq_len, dqkv, B_c, softmax_scale,
                                    s_qi, s_kj, s_vj, s_score_buf, s_lm, s_s_score, s_o);
        }
        CP_ASYNC_WAIT_GROUP(0);
        __syncthreads();

        for (int i = 0; i < STAGE - 1; ++i)
        {
            int j_compute = main_loop_iters + i;
            if (j_compute >= T_c)
                break;
            int compute_stage_idx = j_compute % STAGE;
            compute_attention_block(j_compute, compute_stage_idx, cache_length, start_idx, total_seq_len, dqkv, B_c,
                                    softmax_scale, s_qi, s_kj, s_vj, s_score_buf, s_lm, s_s_score, s_o);
        }

        if (threadIdx.y == 0)
        {
            int out_offset = head_id * (dqkv + 2);
            float &global_m = s_lm[0];
            float &global_l = s_lm[1];
            for (int i = d_tid; i < DQKV_VALUE; i += blockDim.x)
            {
                att_output[out_offset + i] = static_cast<T>(s_o[i]);
            }
            if (d_tid == 0)
            {
                att_output[out_offset + dqkv] = static_cast<T>(global_m);
                att_output[out_offset + dqkv + 1] = static_cast<T>(global_l);
            }
        }
    }

    template <typename T>
    void flash_attention_graph_fixed(Tensor<T> &Q, const Tensor<T> &total_K, const Tensor<T> &total_V, T **d_output_ptrs,
                                     int *d_segment_info, int n_kv_heads, hipStream_t stream, int *pingpong_index)
    {
        int dqkv = Q.sizes()[2];
        if (dqkv != DQKV_VALUE)
        {
            throw std::runtime_error("dqkv does not match the predefined value");
        }

        float softmax_scale = 1.0f / sqrtf(static_cast<float>(dqkv));
        int n_q_h = Q.sizes()[1];
        int n_groups = n_q_h / n_kv_heads;
        int B_r = 1;
        int T_r = 1;
        int B_c = B_C_VALUE;

        const int FIXED_BRANCHES = 3;
        dim3 grid(n_q_h, FIXED_BRANCHES);
        dim3 block(32, B_c);

        constexpr int STAGE = 2;
        size_t smem_size = (DQKV_VALUE * sizeof(float)) +                     // s_qi
                           (STAGE * B_C_VALUE * DQKV_VALUE * sizeof(T)) * 2 + // s_kj, s_vj
                           (B_C_VALUE * sizeof(float)) +                      // s_score_buf
                           (2 * sizeof(float)) +                              // s_lm
                           (B_C_VALUE * sizeof(float)) +                      // s_s_score
                           (DQKV_VALUE * sizeof(float));                      // s_o

        flash_attention_kernel_graph_fixed<T, STAGE><<<grid, block, smem_size, stream>>>(
            Q.data_ptr(), total_K.data_ptr(), total_V.data_ptr(), d_output_ptrs, d_segment_info, n_q_h, n_kv_heads, dqkv,
            B_c, B_r, n_groups, T_r, static_cast<T>(softmax_scale), pingpong_index);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            throw std::runtime_error("CUDA error in flash_attention_graph_fixed: " + std::string(hipGetErrorString(err)));
        }
    }

    template void flash_attention_graph_fixed<float>(Tensor<float> &Q, const Tensor<float> &total_K,
                                                     const Tensor<float> &total_V, float **d_output_ptrs,
                                                     int *d_segment_info, int n_kv_heads, hipStream_t stream,
                                                     int *pingpong_index);

    template void flash_attention_graph_fixed<__hip_bfloat16>(Tensor<__hip_bfloat16> &Q, const Tensor<__hip_bfloat16> &total_K,
                                                             const Tensor<__hip_bfloat16> &total_V,
                                                             __hip_bfloat16 **d_output_ptrs, int *d_segment_info,
                                                             int n_kv_heads, hipStream_t stream, int *pingpong_index);

} // namespace cuda_OP

#endif
#ifndef MULTIST
#include <hipblas.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <math.h>

#include <algorithm>
#include <cstdio>
#include <cstring>
#include <iostream>
#include <stdexcept>
#include <vector>

#include "cudaOP.cuh"

#define DQKV_VALUE 128
#define B_C_VALUE 16
// #define MAX_BRANCHES 5

constexpr int WARP_SIZE = 32;

namespace cuda_OP
{

    // CUDA图优化版本的flash attention kernel
    // 直接从连续的KV缓存中读取数据，仿照flash_attention_variable的模式
    template <typename T>
    __global__ void flash_attention_kernel_graph_fixed(T *q,
                                                       const T *total_k, // 连续的K缓存 [total_seq_len, n_kv_h, dqkv]
                                                       const T *total_v, // 连续的V缓存 [total_seq_len, n_kv_h, dqkv]
                                                       T **output_ptrs,  // 固定的输出指针数组
                                                       int *segment_info, int n_q_h, int n_kv_h, int dqkv, int B_c, int B_r,
                                                       int n_groups, int T_r, float softmax_scale, int *pingpong_index)
    {
        // 从设备内存读取分段信息
        int total_seq_len = segment_info[*pingpong_index];
        // segment_info[1] (active_branches) 已经无用，始终使用固定3分支

        // 固定使用3分支模式，blockIdx.y就是分支索引(0,1,2)
        const int FIXED_BRANCHES = 3;

        // 检查分支ID是否有效
        if (blockIdx.y >= FIXED_BRANCHES)
            return;

        int branches_needed = FIXED_BRANCHES;
        int tokens_per_branch = (total_seq_len + branches_needed - 1) / branches_needed;

        int start_idx, end_idx;
        if (blockIdx.y == 0)
        {
            start_idx = 0;
            end_idx = min(tokens_per_branch, total_seq_len);
        }
        else if (blockIdx.y == 1)
        {
            start_idx = tokens_per_branch;
            end_idx = min(2 * tokens_per_branch, total_seq_len);
        }
        else
        { // blockIdx.y == 2
            start_idx = 2 * tokens_per_branch;
            end_idx = total_seq_len;
        }

        int cache_length = end_idx - start_idx;

        // 如果分支长度为0，直接退出
        if (cache_length <= 0)
            return;

        int T_c = (cache_length + B_c - 1) / B_c;
        T *att_output = output_ptrs[blockIdx.y];

        // 验证参数
        if (dqkv != DQKV_VALUE || B_c != B_C_VALUE)
            return;

        __shared__ float s_qi[DQKV_VALUE];
        __shared__ T s_vj[B_C_VALUE * DQKV_VALUE];
        __shared__ float s_score_buf[B_C_VALUE];
        __shared__ float s_lm[2];
        __shared__ float s_s_score[B_C_VALUE];
        __shared__ float s_o[DQKV_VALUE];

        const int d_tid = threadIdx.x;
        const int token_tid = threadIdx.y;
        const int head_id = blockIdx.x;
        const int q_offset = head_id * dqkv;
        const int kv_head = head_id / n_groups;
        const int tid = blockDim.x * threadIdx.y + threadIdx.x;
        const int num_threads = blockDim.x * blockDim.y;
        for (int i = tid; i < dqkv; i += num_threads)
        {
            s_qi[i] = static_cast<float>(q[q_offset + i]);
        }

        __syncthreads(); // Synchronize to ensure s_qi is fully loaded before use.

        constexpr int vec_unit = 16 / sizeof(T);
        Vec<T, vec_unit> vk, vv; // vq is no longer needed here
        const int vecCount = dqkv / vec_unit;

        float &global_m = s_lm[0];
        float &global_l = s_lm[1];

        // 遍历 KV 分块
        for (int j = 0; j < T_c; ++j)
        {
            int token_index = j * B_c + token_tid;
            bool valid = (token_index < cache_length);
            float local_score = 0.0f;

            for (int i = d_tid; i < vecCount; i += blockDim.x)
            {

                int absolute_token_idx = start_idx + token_index;

                if (valid && absolute_token_idx < total_seq_len)
                {
                    int index = (absolute_token_idx * n_kv_h + kv_head) * dqkv + i * vec_unit;
                    vk.f4 = *reinterpret_cast<const float4 *>(&total_k[index]);
                    vv.f4 = *reinterpret_cast<const float4 *>(&total_v[index]);
                    *reinterpret_cast<float4 *>(&s_vj[token_tid * DQKV_VALUE + i * vec_unit]) = vv.f4;
#pragma unroll
                    for (int l = 0; l < vec_unit; l++)
                    {
                        float k_val = static_cast<float>(vk.t[l]);

                        local_score += s_qi[i * vec_unit + l] * k_val;
                    }
                }
                else
                {
#pragma unroll
                    // for (int l = 0; l < vec_unit; l++)
                    // {
                    //     s_vj[token_tid * DQKV_VALUE + i * vec_unit + l] = 0.0f;
                    // }
                    *reinterpret_cast<float4 *>(&s_vj[token_tid * DQKV_VALUE + i * vec_unit]) = {0.0f, 0.0f, 0.0f, 0.0f};
                }
            }

            __syncthreads();

            // Warp 内归约 QK Score
            int absolute_token_idx = start_idx + token_index;
            bool absolutely_valid = valid && (absolute_token_idx < total_seq_len);

            if (absolutely_valid)
            {
                unsigned int mask = 0xFFFFFFFF;
                for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2)
                {
                    local_score += __shfl_down_sync(mask, local_score, offset);
                }
                if (d_tid == 0)
                {
                    s_score_buf[token_tid] = local_score * (softmax_scale);
                }
            }
            else
            {
                if (d_tid == 0)
                {
                    s_score_buf[token_tid] = -FLT_MAX;
                }
            }
            __syncthreads();

            // Local Softmax
            __shared__ float cur_m_s;
            float warp_val = (d_tid < B_c && threadIdx.y == 0) ? s_score_buf[d_tid] : -FLT_MAX;
            unsigned int mask_max = 0xFFFFFFFF;
            for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2)
            {
                warp_val = fmaxf(warp_val, __shfl_down_sync(mask_max, warp_val, offset));
            }
            if (d_tid == 0 && threadIdx.y == 0)
            {
                cur_m_s = warp_val;
            }
            __syncthreads();
            float cur_m = cur_m_s;

            __shared__ float cur_l_s;
            float warp_val_l = 0.0f;
            if (d_tid < B_c && threadIdx.y == 0)
            {
                float score_val = s_score_buf[d_tid];
                float exp_val = expf(score_val - cur_m);
                s_s_score[d_tid] = exp_val;
                warp_val_l = exp_val;
            }
            else
            {
                warp_val_l = 0.0f;
            }

            __syncthreads();

            // 求和归约
            unsigned int mask_sum = 0xFFFFFFFF;
            for (int offset = WARP_SIZE / 2; offset > 0; offset >>= 1)
            {
                warp_val_l += __shfl_down_sync(mask_sum, warp_val_l, offset);
            }
            if (d_tid == 0 && threadIdx.y == 0)
            {
                cur_l_s = warp_val_l;
            }
            __syncthreads();
            float cur_l = cur_l_s;

            // 计算部分输出
            if (j == 0)
            {
                // 第一个块: 计算并直接写入 s_o
                if (token_tid == 0)
                {
                    for (int k_dim = d_tid; k_dim < DQKV_VALUE; k_dim += blockDim.x)
                    {
                        float current_dim_partial_out = 0.0f;
                        for (int i_tok = 0; i_tok < B_c; ++i_tok)
                        {
                            float exp_score = s_s_score[i_tok];
                            float v_val = static_cast<float>(s_vj[i_tok * DQKV_VALUE + k_dim]);
                            current_dim_partial_out = fmaf(exp_score, v_val, current_dim_partial_out);
                        }
                        s_o[k_dim] = current_dim_partial_out;
                    }
                }
                if (token_tid == 0 && d_tid == 0)
                {
                    global_m = cur_m;
                    global_l = cur_l;
                }
            }
            else
            {
                // 后续块: Online update
                float old_global_m = global_m;
                float old_global_l = global_l;
                float new_global_m = fmaxf(old_global_m, cur_m);
                float exp_old = __expf(old_global_m - new_global_m);
                float exp_cur = __expf(cur_m - new_global_m);

                if (token_tid == 0)
                {
                    for (int k_dim = d_tid; k_dim < DQKV_VALUE; k_dim += blockDim.x)
                    {
                        float current_dim_partial_out = 0.0f;
                        for (int i_tok = 0; i_tok < B_c; ++i_tok)
                        {
                            float exp_score = s_s_score[i_tok];
                            float v_val = static_cast<float>(s_vj[i_tok * DQKV_VALUE + k_dim]);
                            current_dim_partial_out = fmaf(exp_score, v_val, current_dim_partial_out);
                        }
                        float old_out_val = s_o[k_dim];
                        float new_out_val = old_out_val * exp_old + current_dim_partial_out * exp_cur;
                        s_o[k_dim] = new_out_val;
                    }
                }

                if (token_tid == 0 && d_tid == 0)
                {
                    float new_global_l = old_global_l * exp_old + cur_l * exp_cur;
                    global_m = new_global_m;
                    global_l = new_global_l;
                }
            }
            __syncthreads();
        }

        // 写回 att_output
        if (threadIdx.y == 0)
        {
            int out_offset = head_id * (dqkv + 2);
            for (int i = d_tid; i < DQKV_VALUE; i += blockDim.x)
            {
                att_output[out_offset + i] = static_cast<T>(s_o[i]);
            }
            if (d_tid == 0)
            {
                att_output[out_offset + dqkv] = static_cast<T>(global_m);
                att_output[out_offset + dqkv + 1] = static_cast<T>(global_l);
            }
        }
    }

    // CUDA图优化版本：使用固定内存地址和分段信息的flash attention
    template <typename T>
    void flash_attention_graph_fixed(Tensor<T> &Q, const Tensor<T> &total_K, const Tensor<T> &total_V, T **d_output_ptrs,
                                     int *d_segment_info, int n_kv_heads, hipStream_t stream, int *pingpong_index)
    {
        int dqkv = Q.sizes()[2];
        if (dqkv != DQKV_VALUE)
        {
            throw std::runtime_error("dqkv 不匹配预定义的值");
        }

        float softmax_scale = 1.0f / sqrtf(static_cast<float>(dqkv));
        int n_q_h = Q.sizes()[1];
        int n_groups = n_q_h / n_kv_heads;
        int B_r = 1;
        int T_r = 1;
        int B_c = B_C_VALUE;

        // 设置kernel参数 - 强制使用3分支，类似flash_attention.cu的稳定模式
        const int FIXED_BRANCHES = 3;
        dim3 grid(n_q_h, FIXED_BRANCHES);
        dim3 block(32, B_c);

        // 启动kernel
        flash_attention_kernel_graph_fixed<T><<<grid, block, 0, stream>>>(
            Q.data_ptr(), total_K.data_ptr(), total_V.data_ptr(), d_output_ptrs, d_segment_info, n_q_h, n_kv_heads, dqkv,
            B_c, B_r, n_groups, T_r, (softmax_scale), pingpong_index);

        // 检查错误
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            throw std::runtime_error("CUDA error in flash_attention_graph_fixed: " + std::string(hipGetErrorString(err)));
        }
    }

    // 显式模板实例化
    template void flash_attention_graph_fixed<float>(Tensor<float> &Q, const Tensor<float> &total_K,
                                                     const Tensor<float> &total_V, float **d_output_ptrs,
                                                     int *d_segment_info, int n_kv_heads, hipStream_t stream,
                                                     int *pingpong_index);

    template void flash_attention_graph_fixed<__hip_bfloat16>(Tensor<__hip_bfloat16> &Q, const Tensor<__hip_bfloat16> &total_K,
                                                             const Tensor<__hip_bfloat16> &total_V,
                                                             __hip_bfloat16 **d_output_ptrs, int *d_segment_info,
                                                             int n_kv_heads, hipStream_t stream, int *pingpong_index);

} // namespace cuda_OP
#endif