#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_bf16.h>  // 提供 __hip_bfloat16 定义
#include <hip/hip_runtime.h>
#include <float.h>
#include <math.h>

#include <algorithm>  // min
#include <cstdio>     // printf
#include <cstring>    // memcpy
#include <iostream>
#include <stdexcept>
#include <vector>

#include "cudaOP.cuh"  // 假设此头文件定义了 Tensor<T> 等接口

// 预先确定的参数（根据实际情况调整）
#define DQKV_VALUE 128
#define B_C_VALUE 8

// ----------------------------------------------
// bf16 与 float 转换函数
// ----------------------------------------------
__device__ inline float bf16_to_float(__hip_bfloat16 x) {
  unsigned short raw;
  memcpy(&raw, &x, sizeof(raw));
  unsigned int bits = (static_cast<unsigned int>(raw) << 16);
  float f;
  memcpy(&f, &bits, sizeof(f));
  return f;
}

__device__ inline __hip_bfloat16 float_to_bf16(float f) {
  unsigned int bits;
  memcpy(&bits, &f, sizeof(bits));
  unsigned short raw = static_cast<unsigned short>(bits >> 16);
  __hip_bfloat16 h;
  memcpy(&h, &raw, sizeof(h));
  return h;
}

// ----------------------------------------------
// 自定义指数函数 my_exp
// ----------------------------------------------
template <typename T>
__device__ inline T my_exp(T x);

template <>
__device__ inline float my_exp<float>(float x) {
  return expf(x);
}

template <>
__device__ inline double my_exp<double>(double x) {
  return exp(x);
}

template <>
__device__ inline __hip_bfloat16 my_exp<__hip_bfloat16>(__hip_bfloat16 x) {
  float fx = bf16_to_float(x);
  float ef = expf(fx);
  return float_to_bf16(ef);
}

// ----------------------------------------------
// 自定义最大值函数 my_fmax
// ----------------------------------------------
template <typename T>
__device__ inline T my_fmax(T a, T b);

template <>
__device__ inline float my_fmax<float>(float a, float b) {
  return fmaxf(a, b);
}

template <>
__device__ inline __hip_bfloat16 my_fmax<__hip_bfloat16>(__hip_bfloat16 a,
                                                       __hip_bfloat16 b) {
  float fa = bf16_to_float(a);
  float fb = bf16_to_float(b);
  float fm = fmaxf(fa, fb);
  return float_to_bf16(fm);
}

namespace cuda_OP {

// 仅保留float计算版本。BF16计算精度不足。
template <typename T>
__global__ void flash_attention_kernel_v2(T* q, const T* k, const T* v,
                                          T* att_output, int n_q_h,
                                          int cache_length, int n_kv_h,
                                          int dqkv, int B_c, int B_r,
                                          int n_groups, int T_r, int T_c,
                                          T softmax_scale) {
  // 检查预设参数是否一致
  if (dqkv != DQKV_VALUE || B_c != B_C_VALUE) return;

  // 共享内存均采用 float 类型
  __shared__ float s_qi[DQKV_VALUE];  // 当前 head 的 Query（float）
  __shared__ float s_kj[B_C_VALUE * DQKV_VALUE];  // 当前 chunk 的 Key（float）
  __shared__ float
      s_vj[B_C_VALUE * DQKV_VALUE];         // 当前 chunk 的 Value（float）
  __shared__ float s_score_buf[B_C_VALUE];  // 每个 token 的得分
  __shared__ float s_tmp[DQKV_VALUE * B_C_VALUE];  // 临时归约数组
  // s_lm[0 ~ B_C_VALUE-1] 用于归约；s_lm[B_C_VALUE]
  // 存全局最大值；s_lm[B_C_VALUE+1] 存全局归一化因子
  __shared__ float s_lm[2];
  __shared__ float s_max_local[B_C_VALUE];  // 局部最大值归约缓存
  __shared__ float s_exp_local[B_C_VALUE];  // 局部指数和归约缓存

  // 线程内变量
  const int d_tid = threadIdx.x;  // 特征维度内线程索引，[0, dqkv)
  const int token_tid = threadIdx.y;  // 当前 chunk 内 token 线程索引，[0, B_c)
  const int head_id = blockIdx.x;
  const int q_offset = head_id * dqkv;
  const int kv_head = head_id / n_groups;  // KV head 索引

  // --------------------------
  // 1. 初始化输出：转换 0.0f 到类型 T
  // --------------------------
  for (int i = d_tid; i < dqkv; i += blockDim.x) {
    att_output[q_offset + i] = static_cast<T>(0.0f);
  }
  __syncthreads();

  // --------------------------
  // 2. 加载当前 head 的 Query 到共享内存（float）
  // --------------------------
  if (token_tid == 0) {
    for (int i = d_tid; i < dqkv; i += blockDim.x) {
      s_qi[i] = static_cast<float>(q[q_offset + i]);
    }
  }
  __syncthreads();

  // s_lm[B_C_VALUE] 存储全局最大值，s_lm[B_C_VALUE+1] 存储全局归一化因子
  float& global_m = s_lm[0];
  float& global_l = s_lm[1];

  // --------------------------
  // 3. 遍历每个 KV 分块（chunk）
  // --------------------------
  for (int j = 0; j < T_c; ++j) {
    int token_index = j * B_c + token_tid;
    bool valid = (token_index < cache_length);

    // 3.1 加载当前分块的 Key 与 Value（无效位置填 0）
    for (int i = d_tid; i < dqkv; i += blockDim.x) {
      if (valid) {
        int index = (token_index * n_kv_h + kv_head) * dqkv + i;
        s_kj[token_tid * dqkv + i] = static_cast<float>(k[index]);
        s_vj[token_tid * dqkv + i] = static_cast<float>(v[index]);
      } else {
        s_kj[token_tid * dqkv + i] = 0.0f;
        s_vj[token_tid * dqkv + i] = 0.0f;
      }
    }
    __syncthreads();

    // 3.2 计算 Query 与每个 Key 的点积得分（float）
    float score = 0.0f;
    if (valid) {
      for (int i = d_tid; i < dqkv; i += blockDim.x) {
        score += s_qi[i] * s_kj[token_tid * dqkv + i];
      }
      // 每个线程计算的局部得分归约到同一 token 上
      int index = token_tid * dqkv + d_tid;
      s_tmp[index] = score;
      __syncthreads();
      for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (d_tid < stride) {
          s_tmp[index] += s_tmp[index + stride];
        }
        __syncthreads();
      }
      if (d_tid == 0) {
        // 应用 softmax 缩放因子
        s_score_buf[token_tid] =
            s_tmp[token_tid * dqkv] * static_cast<float>(softmax_scale);
      }
    } else {
      if (d_tid == 0) {
        s_score_buf[token_tid] = -FLT_MAX;
      }
    }
    __syncthreads();

    // 3.3 计算当前分块 softmax 归一化参数
    // 3.3.1 并行归约求最大值 cur_m
    if (d_tid == 0) {
      s_max_local[token_tid] = s_score_buf[token_tid];
    }
    __syncthreads();
    for (int stride = B_c / 2; stride > 0; stride >>= 1) {
      if (d_tid == 0 && token_tid < stride) {
        s_max_local[token_tid] =
            fmaxf(s_max_local[token_tid], s_max_local[token_tid + stride]);
      }
      __syncthreads();
    }
    float cur_m = s_max_local[0];
    __syncthreads();

    // 3.3.2 并行归约求指数和 cur_l
    if (d_tid == 0) {
      s_exp_local[token_tid] = expf(s_score_buf[token_tid] - cur_m);
    }
    __syncthreads();
    for (int stride = B_c / 2; stride > 0; stride >>= 1) {
      if (d_tid == 0 && token_tid < stride) {
        s_exp_local[token_tid] += s_exp_local[token_tid + stride];
      }
      __syncthreads();
    }
    float cur_l = s_exp_local[0];
    __syncthreads();

    // 3.4 计算当前分块部分输出 partial_out（float 累积）
    float partial_out = 0.0f;
    for (int i = 0; i < B_c; ++i) {
      float weight = expf(s_score_buf[i] - cur_m) / cur_l;
      partial_out += weight * s_vj[i * dqkv + d_tid];
    }

    // 3.5 更新全局 softmax 参数与输出（递归归一化）
    if (j == 0) {
      if (token_tid == 0) {
        att_output[q_offset + d_tid] = static_cast<T>(partial_out);
      }
      if (token_tid == 0 && d_tid == 0) {
        global_m = cur_m;
        global_l = cur_l;
      }
      __syncthreads();
    } else {
      float new_global_m = fmaxf(global_m, cur_m);
      float exp_old = expf(global_m - new_global_m);
      float exp_cur = expf(cur_m - new_global_m);
      float new_global_l = global_l * exp_old + cur_l * exp_cur;
      float old_out = static_cast<float>(att_output[q_offset + d_tid]);
      float new_out =
          (global_l * exp_old * old_out + cur_l * exp_cur * partial_out) /
          new_global_l;
      if (token_tid == 0) {
        att_output[q_offset + d_tid] = static_cast<T>(new_out);
      }
      if (token_tid == 0 && d_tid == 0) {
        global_m = new_global_m;
        global_l = new_global_l;
      }
      __syncthreads();
    }
  }  // end for each KV chunk
}

// -------------------------------
// host 端调用：设置 grid/block、使用静态共享内存（因此 shmem_bytes
// 设为0），并发起 kernel 调用
template <typename T>
void flash_attention(Tensor<T>& Q, const Tensor<T>& K, const Tensor<T>& V,
                     Tensor<T>& att_output) {
  int dqkv = K.sizes()[2];  // 每个 head 内维度
  if (dqkv != DQKV_VALUE) {
    throw std::runtime_error("dqkv 不匹配预定义的值");
  }
  float softmax_scale = 1.0f / sqrtf(static_cast<float>(dqkv));
  int n_q_h = Q.sizes()[1];         // query head 数
  int cache_length = K.sizes()[0];  // 总的 kv token 数
  int n_kv_h = K.sizes()[1];
  int n_groups = n_q_h / n_kv_h;  // GQA 中的分组数

  // decode 模式下 query 长度为 1
  int B_r = 1;
  int T_r = 1;

  // 每个 chunk 读取的 kv token 数（预设为偶数 B_C_VALUE）
  int B_c = B_C_VALUE;
  int T_c = (cache_length + B_c - 1) / B_c;

  // 每个 block 处理一个 query head
  dim3 grid(n_q_h);
  int threads_x = dqkv;  // dqkv = DQKV_VALUE
  int threads_y = B_c;   // B_c = B_C_VALUE
  dim3 block(threads_x, threads_y);

  // 使用静态共享内存，故 shmem_bytes = 0
  flash_attention_kernel_v2<T><<<grid, block, 0>>>(
      Q.data_ptr(), K.data_ptr(), V.data_ptr(), att_output.data_ptr(), n_q_h,
      cache_length, n_kv_h, dqkv, B_c, B_r, n_groups, T_r, T_c,
      static_cast<T>(softmax_scale));
}

// 显式实例化
template void flash_attention<float>(Tensor<float>&, const Tensor<float>&,
                                     const Tensor<float>&, Tensor<float>&);
template void flash_attention<nvbf16>(Tensor<nvbf16>&, const Tensor<nvbf16>&,
                                      const Tensor<nvbf16>&, Tensor<nvbf16>&);

}  // namespace cuda_OP
